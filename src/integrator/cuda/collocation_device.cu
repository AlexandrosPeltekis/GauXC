#include "collocation_kernels.hpp"
#include <gauxc/util/div_ceil.hpp>


namespace GauXC      {
namespace integrator {
namespace cuda       {

template <typename T>
void eval_collocation(
  size_t          nshells,
  size_t          nbf,
  size_t          npts,
  const Shell<T>* shells_device,
  const size_t*   offs_device,
  const T*        pts_device,
  T*              eval_device,
  hipStream_t    stream
) {


  dim3 threads(32, 32, 1);
  dim3 blocks( util::div_ceil( npts,    threads.x ),
               util::div_ceil( nshells, threads.y ) );

  collocation_device_kernel<T>
    <<<blocks, threads, 0, stream>>>
    ( nshells, nbf, npts, shells_device, offs_device,
      pts_device, eval_device );

}
 
template             
void eval_collocation(
  size_t               nshells,
  size_t               nbf,
  size_t               npts,
  const Shell<double>* shells_device,
  const size_t*        offs_device,
  const double*        pts_device,
  double*              eval_device,
  hipStream_t         stream
);









template <typename T>
void eval_collocation(
  size_t          nshells,
  size_t          nbf,
  size_t          npts,
  const Shell<T>* shells_device,
  const size_t*   mask_device,
  const size_t*   offs_device,
  const T*        pts_device,
  T*              eval_device,
  hipStream_t    stream
) {

  dim3 threads(32, 32, 1);
  dim3 blocks( util::div_ceil( npts,    threads.x ),
               util::div_ceil( nshells, threads.y ) );

  collocation_device_kernel<T>
    <<<blocks, threads, 0, stream>>>
    ( nshells, nbf, npts, shells_device, mask_device,
      offs_device, pts_device, eval_device );

}
 
template             
void eval_collocation(
  size_t               nshells,
  size_t               nbf,
  size_t               npts,
  const Shell<double>* shells_device,
  const size_t*        mask_device,
  const size_t*        offs_device,
  const double*        pts_device,
  double*              eval_device,
  hipStream_t         stream
);


















template <typename T>
void eval_collocation_deriv1(
  size_t          nshells,
  size_t          nbf,
  size_t          npts,
  const Shell<T>* shells_device,
  const size_t*   offs_device,
  const T*        pts_device,
  T*              eval_device,
  T*              deval_device_x,
  T*              deval_device_y,
  T*              deval_device_z,
  hipStream_t    stream
) {

  dim3 threads(32, 32, 1);
  dim3 blocks( util::div_ceil( npts,    threads.x ),
               util::div_ceil( nshells, threads.y ) );

  collocation_device_kernel_deriv1<T>
    <<<blocks, threads, 0, stream>>>
    ( nshells, nbf, npts, shells_device, offs_device,
      pts_device, eval_device, deval_device_x, deval_device_y,
      deval_device_z );

}

template
void eval_collocation_deriv1(
  size_t               nshells,
  size_t               nbf,
  size_t               npts,
  const Shell<double>* shells_device,
  const size_t*        offs_device,
  const double*        pts_device,
  double*              eval_device,
  double*              deval_device_x,
  double*              deval_device_y,
  double*              deval_device_z,
  hipStream_t         stream
);
















template <typename T>
void eval_collocation_deriv1(
  size_t          nshells,
  size_t          nbf,
  size_t          npts,
  const Shell<T>* shells_device,
  const size_t*   mask_device,
  const size_t*   offs_device,
  const T*        pts_device,
  T*              eval_device,
  T*              deval_device_x,
  T*              deval_device_y,
  T*              deval_device_z,
  hipStream_t    stream
) {

  dim3 threads(32, 32, 1);
  dim3 blocks( util::div_ceil( npts,    threads.x ),
               util::div_ceil( nshells, threads.y ) );

  collocation_device_kernel_deriv1<T>
    <<<blocks, threads, 0, stream>>>
    ( nshells, nbf, npts, shells_device, mask_device, offs_device,
      pts_device, eval_device, deval_device_x, deval_device_y,
      deval_device_z );

}

template
void eval_collocation_deriv1(
  size_t               nshells,
  size_t               nbf,
  size_t               npts,
  const Shell<double>* shells_device,
  const size_t*        mask_device,
  const size_t*        offs_device,
  const double*        pts_device,
  double*              eval_device,
  double*              deval_device_x,
  double*              deval_device_y,
  double*              deval_device_z,
  hipStream_t         stream
);


} // namespace cuda
} // namespace integrator
} // namespace GauXC
