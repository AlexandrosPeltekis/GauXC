#include "hip/hip_runtime.h"
#include <math.h>
#include "../include/gpu/chebyshev_boys_computation.hpp"
#include "config_obara_saika.hpp"
#include "integral_0.hu"

namespace XGPU {
  __inline__ __device__ void dev_integral_0_driver(size_t npts,
				 double *_points_x,
				 double *_points_y,
				 double *_points_z,
         shell_pair* sp,
				 double *Xi,
				 int ldX,
				 double *Gi,
				 int ldG, 
				 double *weights,
				 double *boys_table) {
    __shared__ double temp[128 * 1];
    const auto nprim_pairs = sp->nprim_pairs();
    const auto prim_pairs  = sp->prim_pairs();
    
    for(size_t p_outer = blockIdx.x * blockDim.x; p_outer < npts; p_outer += gridDim.x * blockDim.x) {
      double *_point_outer_x = (_points_x + p_outer);
      double *_point_outer_y = (_points_y + p_outer);
      double *_point_outer_z = (_points_z + p_outer);
      
      size_t p_inner = (threadIdx.x < (npts - p_outer)) ? threadIdx.x : (npts - p_outer);

      for(int i = 0; i < 1; ++i) SCALAR_STORE((temp + i * blockDim.x + threadIdx.x), SCALAR_ZERO());

      for(int ij = 0; ij < nprim_pairs; ++ij) {
	double RHO = prim_pairs[ij].gamma;

	double xA = prim_pairs[ij].P.x;
	double yA = prim_pairs[ij].P.y;
	double zA = prim_pairs[ij].P.z;
	
	double eval = prim_pairs[ij].K_coeff_prod;

	// Evaluate T Values
	SCALAR_TYPE xC = SCALAR_LOAD((_point_outer_x + p_inner));
	SCALAR_TYPE yC = SCALAR_LOAD((_point_outer_y + p_inner));
	SCALAR_TYPE zC = SCALAR_LOAD((_point_outer_z + p_inner));

	SCALAR_TYPE X_PC = SCALAR_SUB(xA, xC);
	SCALAR_TYPE Y_PC = SCALAR_SUB(yA, yC);
	SCALAR_TYPE Z_PC = SCALAR_SUB(zA, zC);

	X_PC = SCALAR_MUL(X_PC, X_PC);
	X_PC = SCALAR_FMA(Y_PC, Y_PC, X_PC);
	X_PC = SCALAR_FMA(Z_PC, Z_PC, X_PC);
	SCALAR_TYPE TVAL = SCALAR_MUL(RHO, X_PC);

	SCALAR_TYPE t00, TVAL_inv_e;

	// Evaluate Boys function
	boys_element<0>(&TVAL, &TVAL_inv_e, &t00, boys_table);
	
	// Evaluate VRR Buffer
	SCALAR_TYPE tx;

	t00 = SCALAR_MUL(eval, t00);
	tx = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t00);
	SCALAR_STORE((temp + 0 * blockDim.x + threadIdx.x), tx);
      }

      if(threadIdx.x < npts - p_outer) {
	double *Xik = (Xi + p_outer + p_inner);
	double *Gik = (Gi + p_outer + p_inner);

	SCALAR_TYPE tx, wg, xik, gik;
	tx  = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	wg  = SCALAR_LOAD((weights + p_outer + p_inner));

	xik = SCALAR_LOAD((Xik + 0 * ldX));
	gik = SCALAR_LOAD((Gik + 0 * ldG));

	tx = SCALAR_MUL(tx, wg);
	gik = SCALAR_FMA(tx, xik, gik);
	SCALAR_STORE((Gik + 0 * ldG), gik);
      }
    }
  }

  __global__ void dev_integral_0(size_t npts,
				   double *points_x,
				   double *points_y,
				   double *points_z,
           shell_pair* sp,
				   double *Xi,
				   int ldX,
				   double *Gi,
				   int ldG, 
				   double *weights, 
				   double *boys_table) {
    dev_integral_0_driver( npts, points_x, points_y, points_z, sp, Xi, ldX,
      Gi, ldG, weights, boys_table );
  }

  void integral_0(size_t npts,
		  double *_points_x,	
		  double *_points_y,	
		  double *_points_z,	
      shell_pair* sp,
		  double *Xi,
		  int ldX,
		  double *Gi,
		  int ldG, 
		  double *weights,
		  double *boys_table,
      hipStream_t stream) {
    dev_integral_0<<<320, 128, 0, stream>>>(npts,
				 _points_x,
				 _points_y,
				 _points_z,
         sp,
				 Xi,
				 ldX,
				 Gi,
				 ldG, 
				 weights, 
				 boys_table);
  }

  __global__ void dev_integral_0_batched(
           const GauXC::ShellPairToTaskDevice* sp2task,
           GauXC::XCDeviceTask*                device_tasks,
				   double *boys_table) {

    const int ntask = sp2task->ntask;
    for( int i_task = blockIdx.y; i_task < ntask; i_task += gridDim.y ) {
    
      const auto iT = sp2task->task_idx_device[i_task];
      const auto* task  = device_tasks + iT;
      const auto  npts  = task->npts;

      const auto  i_off = sp2task->task_shell_off_row_device[i_task]*npts;

      dev_integral_0_driver( 
        npts,
        task->points_x,
        task->points_y,
        task->points_z,
        sp2task->shell_pair_device,
        task->fmat + i_off,
        npts,
        task->gmat + i_off,
        npts,
        task->weights, boys_table );
    }

  }


  void integral_0_batched(size_t ntask_sp,
        const GauXC::ShellPairToTaskDevice* sp2task,
        GauXC::XCDeviceTask*                device_tasks,
		    double *boys_table,
        hipStream_t stream) {

    int nthreads = 128;
    int nblocks_x = 160;
    int nblocks_y = ntask_sp;
    dim3 nblocks(nblocks_x, nblocks_y);
    dev_integral_0_batched<<<nblocks,nthreads,0,stream>>>(
      sp2task, device_tasks, boys_table );

  }

#define K0_WARPSIZE 32
#define K0_NUMWARPS 8
#define K0_NUMTHREADS (K0_WARPSIZE * K0_NUMWARPS)
#define K0_USESHARED 1
#define K0_MAX_PRIMPAIRS 32

__inline__ __device__ void dev_integral_0_task(
  const int i,
  const int npts,
  const int nprim_pairs,
  // Point data
  double4 (&s_task_data)[K0_NUMTHREADS],
  // Shell Pair Data
  const shell_pair* sp,
  // Output Data
  const double *Xi,
  int ldX,
  double *Gi,
  int ldG, 
  // Other
  const double *boys_table) {


  const auto& prim_pairs = sp->prim_pairs();

  const int laneId = threadIdx.x % K0_WARPSIZE;

#if K0_USESHARED
  // Load Primpairs to shared
  __shared__ GauXC::PrimitivePair<double> s_prim_pairs[K0_NUMWARPS][K0_MAX_PRIMPAIRS];

  const int warpId = (threadIdx.x / K0_WARPSIZE);
  const int32_t* src = (int32_t*) &(prim_pairs[0]);
  int32_t* dst = (int32_t*) &(s_prim_pairs[warpId][0]);

  for (int i = laneId; i < nprim_pairs * sizeof(GauXC::PrimitivePair<double>) / sizeof(int32_t); i+=K0_WARPSIZE) {
    dst[i] = src[i]; 
  }
  __syncwarp();
#endif

  // Loop over points in shared in batches of 32
  for (int i = 0; i < K0_NUMTHREADS / K0_WARPSIZE; i++) {
    double temp = SCALAR_ZERO();

    const int pointIndex = i * K0_WARPSIZE + laneId;

    if (pointIndex < npts) {

      const double point_x = s_task_data[pointIndex].x;
      const double point_y = s_task_data[pointIndex].y;
      const double point_z = s_task_data[pointIndex].z;
      const double weight = s_task_data[pointIndex].w;

      for (int ij = 0; ij < nprim_pairs; ij++) {
#if K0_USESHARED
        double RHO = s_prim_pairs[warpId][ij].gamma;
        double xP = s_prim_pairs[warpId][ij].P.x;
        double yP = s_prim_pairs[warpId][ij].P.y;
        double zP = s_prim_pairs[warpId][ij].P.z;
        double eval = s_prim_pairs[warpId][ij].K_coeff_prod;
#else
        double RHO = prim_pairs[ij].gamma;
        double xP = prim_pairs[ij].P.x;
        double yP = prim_pairs[ij].P.y;
        double zP = prim_pairs[ij].P.z;
        double eval = prim_pairs[ij].K_coeff_prod;
#endif
      
        // Evaluate T Values
        const SCALAR_TYPE X_PC = SCALAR_SUB(xP, point_x);
        const SCALAR_TYPE Y_PC = SCALAR_SUB(yP, point_y);
        const SCALAR_TYPE Z_PC = SCALAR_SUB(zP, point_z);
      
        SCALAR_TYPE TVAL = SCALAR_MUL(X_PC, X_PC);
        TVAL = SCALAR_FMA(Y_PC, Y_PC, TVAL);
        TVAL = SCALAR_FMA(Z_PC, Z_PC, TVAL);
        TVAL = SCALAR_MUL(RHO, TVAL);
      
        // Evaluate VRR Buffer
        const SCALAR_TYPE t00 = boys_element_0(TVAL);
        temp = SCALAR_FMA( eval, t00, temp );
      }

      // Output
     // if (abs(temp) > 1e-12)
      {
        const double * __restrict__ Xik = (Xi + pointIndex);
        double * __restrict__ Gik = (Gi + pointIndex);

        SCALAR_TYPE const_value_v = weight;
      
        double const_value, X_ABp, Y_ABp, Z_ABp, comb_m_i, comb_n_j, comb_p_k;
        SCALAR_TYPE const_value_w;
        SCALAR_TYPE tx, ty, tz, tw, t0;
      
        X_ABp = 1.0; comb_m_i = 1.0;
        Y_ABp = 1.0; comb_n_j = 1.0;
        Z_ABp = 1.0; comb_p_k = 1.0;
        const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
        const_value_w = SCALAR_MUL(const_value_v, const_value);
        tx = SCALAR_LOAD(Xik);
        t0 = SCALAR_MUL(temp, const_value_w);
        tw = SCALAR_MUL(tx, t0);
        atomicAdd(Gik, tw);
      }
    }
  }
  __syncwarp();
}

__global__ void 
__launch_bounds__(K0_NUMTHREADS, 1)
dev_integral_0_task_batched(
  int ntask, int nsubtask,
  GauXC::XCDeviceTask*                device_tasks,
  const GauXC::TaskToShellPairDevice* task2sp,
  const int4* subtasks,
  const int32_t* nprim_pairs_device,
  shell_pair** sp_ptr_device,
  double *boys_table) {

  __shared__ double4 s_task_data[K0_NUMTHREADS];

  const int warpId = threadIdx.x / K0_WARPSIZE;
  
  const int i_subtask = blockIdx.x;
  const int i_task = subtasks[i_subtask].x;
  const int point_start = subtasks[i_subtask].y;
  const int point_end = subtasks[i_subtask].z;
  const int point_count = point_end - point_start;

  const auto* task = device_tasks + i_task;

  const int npts = task->npts;

  const auto* points_x = task->points_x;
  const auto* points_y = task->points_y;
  const auto* points_z = task->points_z;
  const auto* weights = task->weights;

  const auto nsp = task2sp[i_task].nsp;


  const int npts_block = (point_count + blockDim.x - 1) / blockDim.x;

  for (int i_block = 0; i_block < npts_block; i_block++) {
    const int i = point_start + i_block * blockDim.x;

    // load point into registers
    const double point_x = points_x[i + threadIdx.x];
    const double point_y = points_y[i + threadIdx.x];
    const double point_z = points_z[i + threadIdx.x];
    const double weight = weights[i + threadIdx.x];

    s_task_data[threadIdx.x].x = point_x;
    s_task_data[threadIdx.x].y = point_y;
    s_task_data[threadIdx.x].z = point_z;
    s_task_data[threadIdx.x].w = weight;
    __syncthreads();

    for (int j = K0_NUMWARPS*blockIdx.y+warpId; j < nsp; j+=K0_NUMWARPS*gridDim.y) {
      const auto i_off = task2sp[i_task].task_shell_off_row_device[j];

      const auto index =  task2sp[i_task].shell_pair_linear_idx_device[j];
      const auto* sp = sp_ptr_device[index];
      const auto nprim_pairs = nprim_pairs_device[index];

      dev_integral_0_task(
        i, point_count, nprim_pairs,
        s_task_data,
        sp,
        task->fmat + i_off + i,
        npts,
        task->gmat + i_off + i,
        npts,
        boys_table);
    }
    __syncthreads();
  }
}

  void integral_0_task_batched(
    size_t ntasks, size_t nsubtask,
    size_t max_nsp,
    GauXC::XCDeviceTask*                device_tasks,
    const GauXC::TaskToShellPairDevice* task2sp,
    const std::array<int32_t, 4>*  subtasks,
    const int32_t* nprim_pairs_device,
    shell_pair** sp_ptr_device,
    double* sp_X_AB_device,
    double* sp_Y_AB_device,
    double* sp_Z_AB_device,
    double *boys_table,
    hipStream_t stream) {

    size_t xy_max = (1ul << 16) - 1;
    int nthreads = K0_NUMTHREADS;
    int nblocks_x = nsubtask;
    int nblocks_y = 8; //std::min(max_nsp,  xy_max);
    int nblocks_z = 1;
    dim3 nblocks(nblocks_x, nblocks_y, nblocks_z);

    dev_integral_0_task_batched<<<nblocks, nthreads, 0, stream>>>(
      ntasks, nsubtask,
      device_tasks, task2sp, 
      (int4*) subtasks, nprim_pairs_device, sp_ptr_device,
      boys_table );
  }
}
