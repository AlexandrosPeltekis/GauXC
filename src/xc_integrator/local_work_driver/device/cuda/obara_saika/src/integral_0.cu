#include "hip/hip_runtime.h"
#include <math.h>
#include "chebyshev_boys_computation.hpp"
#include "integral_data_types.hpp"
#include "config_obara_saika.hpp"
#include "integral_0.hu"

#define PI 3.14159265358979323846

#define MIN(a,b)			\
  ({ __typeof__ (a) _a = (a);	        \
  __typeof__ (b) _b = (b);		\
  _a < _b ? _a : _b; })

__global__ void integral_0(size_t npts,
                          shell_pair *shpair,
                          double *_points,
                          double *Xi,
                          int ldX,
                          double *Gi,
                          int ldG, 
                          double *weights) {
   __shared__ double *temp;
   for(size_t p_outer = blockIdx.x * blockDim.x; p_outer < npts; p_outer += gridDim.x * blockDim.x) {
      double *_point_outer = (_points + p_outer);

      size_t p_inner = (threadIdx.x < (npts - p_outer)) ? threadIdx.x : (npts - p_outer);

      double xA = shpair[0].rA.x;
      double yA = shpair[0].rA.y;
      double zA = shpair[0].rA.z;

      for(int i = 0; i < 1; ++i) SCALAR_STORE((temp + i * blockDim.x + threadIdx.x), SCALAR_ZERO());

      for(int ij = 0; ij < shpair[0].nprim_pair; ++ij) {
         double RHO = shpair[0].prim_pairs[ij].gamma;

         double eval = shpair[0].prim_pairs[ij].coeff_prod * shpair[0].prim_pairs[ij].K;

         // Evaluate T Values
         SCALAR_TYPE xC = SCALAR_LOAD((_point_outer + p_inner + 0 * npts));
         SCALAR_TYPE yC = SCALAR_LOAD((_point_outer + p_inner + 1 * npts));
         SCALAR_TYPE zC = SCALAR_LOAD((_point_outer + p_inner + 2 * npts));

         SCALAR_TYPE X_PC = SCALAR_SUB(xA, xC);
         SCALAR_TYPE Y_PC = SCALAR_SUB(yA, yC);
         SCALAR_TYPE Z_PC = SCALAR_SUB(zA, zC);

         X_PC = SCALAR_MUL(X_PC, X_PC);
         X_PC = SCALAR_FMA(Y_PC, Y_PC, X_PC);
         X_PC = SCALAR_FMA(Z_PC, Z_PC, X_PC);
         SCALAR_TYPE TVAL = SCALAR_MUL(RHO, X_PC);

         SCALAR_TYPE t00;

         // Evaluate Boys function
         t00 = GauXC::gauxc_boys_element<0>(TVAL);

         // Evaluate VRR Buffer
         SCALAR_TYPE tx;

         t00 = SCALAR_MUL(eval, t00);
         tx = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t00);
         SCALAR_STORE((temp + 0 * blockDim.x + threadIdx.x), tx);
      }

      double *Xik = (Xi + p_outer + p_inner);
      double *Gik = (Gi + p_outer + p_inner);

      SCALAR_TYPE tx, wg, xik, gik;
      tx  = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 0 * ldX));
      gik = SCALAR_LOAD((Gik + 0 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 0 * ldG), gik);
   }
}
