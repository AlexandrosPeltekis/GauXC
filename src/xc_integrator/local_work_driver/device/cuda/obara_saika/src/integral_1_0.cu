#include "hip/hip_runtime.h"
#include <math.h>
#include "../include/gpu/chebyshev_boys_computation.hpp"
#include "config_obara_saika.hpp"
#include "integral_1_0.hu"

namespace XGPU {
  __inline__ __device__ void dev_integral_1_0_driver(size_t npts,
				   double *_points_x,
				   double *_points_y,
				   double *_points_z,
           shell_pair* sp,
				   double *Xi,
				   double *Xj,
				   int ldX,
				   double *Gi,
				   double *Gj,
				   int ldG, 
				   double *weights, 
				   double *boys_table) {
    //__shared__ double temp[128 * 3];
    double temp_0, temp_1, temp_2;
    const auto nprim_pairs = sp->nprim_pairs();
    const auto prim_pairs  = sp->prim_pairs();


    const int npts_int = (int) npts;
    
    for(int p_outer = blockIdx.x * blockDim.x; p_outer < npts_int; p_outer += gridDim.x * blockDim.x) {
      double *_point_outer_x = (_points_x + p_outer);
      double *_point_outer_y = (_points_y + p_outer);
      double *_point_outer_z = (_points_z + p_outer);

      int p_inner = threadIdx.x;
      if (threadIdx.x < npts_int - p_outer) {

      //for(int i = 0; i < 3; ++i) SCALAR_STORE((temp + i * blockDim.x + threadIdx.x), SCALAR_ZERO());
      temp_0 = SCALAR_ZERO();
      temp_1 = SCALAR_ZERO();
      temp_2 = SCALAR_ZERO();

      for(int ij = 0; ij < nprim_pairs; ++ij) {
	double RHO = prim_pairs[ij].gamma;
	double X_PA = prim_pairs[ij].PA.x;
	double Y_PA = prim_pairs[ij].PA.y;
	double Z_PA = prim_pairs[ij].PA.z;

	double xP = prim_pairs[ij].P.x;
	double yP = prim_pairs[ij].P.y;
	double zP = prim_pairs[ij].P.z;

	double eval = prim_pairs[ij].K_coeff_prod;

	// Evaluate T Values
	SCALAR_TYPE xC = SCALAR_LOAD((_point_outer_x + p_inner));
	SCALAR_TYPE yC = SCALAR_LOAD((_point_outer_y + p_inner));
	SCALAR_TYPE zC = SCALAR_LOAD((_point_outer_z + p_inner));

	SCALAR_TYPE X_PC = SCALAR_SUB(xP, xC);
	SCALAR_TYPE Y_PC = SCALAR_SUB(yP, yC);
	SCALAR_TYPE Z_PC = SCALAR_SUB(zP, zC);

	SCALAR_TYPE TVAL = SCALAR_MUL(X_PC, X_PC);
	TVAL = SCALAR_FMA(Y_PC, Y_PC, TVAL);
	TVAL = SCALAR_FMA(Z_PC, Z_PC, TVAL);
	TVAL = SCALAR_MUL(RHO, TVAL);
	
	SCALAR_TYPE t00, t01, TVAL_inv_e;

	// Evaluate Boys function
	boys_element<1>(&TVAL, &TVAL_inv_e, &t01, boys_table);

	// Evaluate VRR Buffer
	SCALAR_TYPE t10, tx;

	t00 = SCALAR_MUL(SCALAR_ADD(SCALAR_MUL(TVAL, t01), TVAL_inv_e), SCALAR_SET1(2.00000000000000000000));

	t00 = SCALAR_MUL(eval, t00);
	t01 = SCALAR_MUL(eval, t01);
	t10 = SCALAR_MUL(X_PA, t00);
	t10 = SCALAR_FNMA(X_PC, t01, t10);
	//tx = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	tx = temp_0;
	tx = SCALAR_ADD(tx, t10);
	//SCALAR_STORE((temp + 0 * blockDim.x + threadIdx.x), tx);
  temp_0 = tx;
	t10 = SCALAR_MUL(Y_PA, t00);
	t10 = SCALAR_FNMA(Y_PC, t01, t10);
	//tx = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	tx = temp_1;
	tx = SCALAR_ADD(tx, t10);
	//SCALAR_STORE((temp + 1 * blockDim.x + threadIdx.x), tx);
  temp_1 = tx;
	t10 = SCALAR_MUL(Z_PA, t00);
	t10 = SCALAR_FNMA(Z_PC, t01, t10);
	//tx = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	tx = temp_2;
	tx = SCALAR_ADD(tx, t10);
	//SCALAR_STORE((temp + 2 * blockDim.x + threadIdx.x), tx);
  temp_2 = tx;
      }

  
      if (abs(temp_0) > 1e-12 || abs(temp_1) > 1e-12 || abs(temp_2) > 1e-12) {
	double *Xik = (Xi + p_outer + p_inner);
	double *Xjk = (Xj + p_outer + p_inner);
	double *Gik = (Gi + p_outer + p_inner);
	double *Gjk = (Gj + p_outer + p_inner);

	SCALAR_TYPE const_value_v = SCALAR_LOAD((weights + p_outer + p_inner));

	double const_value, X_ABp, Y_ABp, Z_ABp, comb_m_i, comb_n_j, comb_p_k;
	SCALAR_TYPE const_value_w;
	SCALAR_TYPE tx, ty, tz, tw, t0, t1, t2;

	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);

  #if 0
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	//t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
  t0 = temp_0;
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	SCALAR_STORE((Gik + 0 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	//t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
  t1 = temp_1;
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	SCALAR_STORE((Gik + 1 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	//t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
  t2 = temp_2;
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	SCALAR_STORE((Gik + 2 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
  #else
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));

	t0 = SCALAR_MUL(temp_0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_MUL(tx, t0);
	atomicAdd((Gik + 0 * ldG), tz);
                                   
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_MUL(temp_1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);

	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_MUL(temp_2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);

	atomicAdd((Gjk + 0 * ldG), tw);
  #endif
      }
      }
    }
  }

  __global__ void dev_integral_1_0(size_t npts,
				   double *points_x,
				   double *points_y,
				   double *points_z,
           shell_pair* sp,
				   double *Xi,
				   double *Xj,
				   int ldX,
				   double *Gi,
				   double *Gj,
				   int ldG, 
				   double *weights, 
				   double *boys_table) {
    dev_integral_1_0_driver( npts, points_x, points_y, points_z, sp, Xi, Xj, ldX,
      Gi, Gj, ldG, weights, boys_table );
  }

    void integral_1_0(size_t npts,
		    double *points_x,
		    double *points_y,
		    double *points_z,
        shell_pair* sp,
		    double *Xi,
		    double *Xj,
		    int ldX,
		    double *Gi,
		    double *Gj,
		    int ldG, 
		    double *weights, 
		  double *boys_table,
      hipStream_t stream) {
      dev_integral_1_0<<<320, 128, 0, stream>>>(npts,
				     points_x,
				     points_y,
				     points_z,
             sp,
				     Xi,
				     Xj,
				     ldX,
				     Gi,
				     Gj,
				     ldG, 
				     weights,
				     boys_table);
    }




  template <bool swap>
  __inline__ __device__ void dev_integral_1_0_batched_driver(
           const GauXC::ShellPairToTaskDevice* sp2task,
           GauXC::XCDeviceTask*                device_tasks,
				   double *boys_table) {

    //if (sp2task->shell_pair_device->nprim_pairs() == 0) return;
    const int ntask = sp2task->ntask;
    #pragma unroll 1
    for( int i_task = blockIdx.y; i_task < ntask; i_task += gridDim.y ) {
    
      const auto iT = sp2task->task_idx_device[i_task];
      const auto* task  = device_tasks + iT;
      const auto  npts  = task->npts;

      int i_off, j_off;
      if constexpr ( swap ) {
        j_off = sp2task->task_shell_off_row_device[i_task]*npts;
        i_off = sp2task->task_shell_off_col_device[i_task]*npts;
      } else {
        i_off = sp2task->task_shell_off_row_device[i_task]*npts;
        j_off = sp2task->task_shell_off_col_device[i_task]*npts;
      }


      dev_integral_1_0_driver( 
        npts,
        task->points_x,
        task->points_y,
        task->points_z,
        sp2task->shell_pair_device,
        task->fmat + i_off,
        task->fmat + j_off,
        npts,
        task->gmat + i_off,
        task->gmat + j_off,
        npts,
        task->weights, boys_table );
    }

  }

  template <bool swap>
  __global__ void dev_integral_1_0_batched(
           const GauXC::ShellPairToTaskDevice* sp2task,
           GauXC::XCDeviceTask*                device_tasks,
				   double *boys_table) {
    dev_integral_1_0_batched_driver<swap>(sp2task,device_tasks,boys_table);
  }


  void integral_1_0_batched(bool swap, size_t ntask_sp,
        const GauXC::ShellPairToTaskDevice* sp2task,
        GauXC::XCDeviceTask*                device_tasks,
		    double *boys_table,
        hipStream_t stream) {

    int nthreads = 128;
    int nblocks_x = 160;
    int nblocks_y = ntask_sp;
    dim3 nblocks(nblocks_x, nblocks_y);

    if(swap)
      dev_integral_1_0_batched<true><<<nblocks,nthreads,0,stream>>>(
        sp2task, device_tasks, boys_table );
    else
      dev_integral_1_0_batched<false><<<nblocks,nthreads,0,stream>>>(
        sp2task, device_tasks, boys_table );

  }

  template <bool swap>
  __global__ void dev_integral_1_0_shell_batched(
           int nsp,
           const GauXC::ShellPairToTaskDevice* sp2task,
           GauXC::XCDeviceTask*                device_tasks,
				   double *boys_table) {
    for(int i = blockIdx.z; i < nsp; i+= gridDim.z ) {
      dev_integral_1_0_batched_driver<swap>(sp2task+i,device_tasks,boys_table);
    }
  }

  void integral_1_0_shell_batched(
        bool swap,
        size_t nsp,
        size_t max_ntask,
        const GauXC::ShellPairToTaskDevice* sp2task,
        GauXC::XCDeviceTask*                device_tasks,
		    double *boys_table,
        hipStream_t stream) {

    size_t xy_max = (1ul << 16) - 1;
    int nthreads = 128;
    int nblocks_x = 1;
    int nblocks_y = std::min(max_ntask, xy_max);
    int nblocks_z = std::min(nsp,  xy_max);
    dim3 nblocks(nblocks_x, nblocks_y, nblocks_z);
    if(swap)
      dev_integral_1_0_shell_batched<true><<<nblocks,nthreads,0,stream>>>(
        nsp, sp2task, device_tasks, boys_table );
    else
      dev_integral_1_0_shell_batched<false><<<nblocks,nthreads,0,stream>>>(
        nsp, sp2task, device_tasks, boys_table );

  }

#define K10_WARPSIZE 32
#define K10_NUMWARPS 8
#define K10_NUMTHREADS (K10_WARPSIZE * K10_NUMWARPS)
#define K10_USESHARED 1
#define K10_MAX_PRIMPAIRS 32

__inline__ __device__ void dev_integral_1_0_task(
  const int i,
  const int npts,
  const int nprim_pairs,
  // Point data
  double4 (&s_task_data)[K10_NUMTHREADS],
  // Shell Pair Data
  const shell_pair* sp,
  // Output Data
  const double *Xi,
  const double *Xj,
  int ldX,
  double *Gi,
  double *Gj,
  int ldG, 
  // Other
  double *boys_table) {

  const int laneId = threadIdx.x % K10_WARPSIZE;

  const auto& prim_pairs = sp->prim_pairs();

#if K10_USESHARED
  // Load Primpairs to shared
  __shared__ GauXC::PrimitivePair<double> s_prim_pairs[K10_NUMWARPS][K10_MAX_PRIMPAIRS];

  const int warpId = (threadIdx.x / K10_WARPSIZE);
  const int32_t* src = (int32_t*) &(prim_pairs[0]);
  int32_t* dst = (int32_t*) &(s_prim_pairs[warpId][0]);

  for (int i = laneId; i < nprim_pairs * sizeof(GauXC::PrimitivePair<double>) / sizeof(int32_t); i+=K10_WARPSIZE) {
    dst[i] = src[i]; 
  }
  __syncwarp();
#endif


  // Loop over points in shared in batches of 32
  for (int i = 0; i < K10_NUMTHREADS / K10_WARPSIZE; i++) {
    double temp_0 = SCALAR_ZERO();
    double temp_1 = SCALAR_ZERO();
    double temp_2 = SCALAR_ZERO();

    const int pointIndex = i * K10_WARPSIZE + laneId;

    if (pointIndex < npts) {
      const double point_x = s_task_data[pointIndex].x;
      const double point_y = s_task_data[pointIndex].y;
      const double point_z = s_task_data[pointIndex].z;
      const double weight = s_task_data[pointIndex].w;

      for(int ij = 0; ij < nprim_pairs; ++ij) {
#if K10_USESHARED
        double RHO = s_prim_pairs[warpId][ij].gamma;
        double X_PA = s_prim_pairs[warpId][ij].PA.x;
        double Y_PA = s_prim_pairs[warpId][ij].PA.y;
        double Z_PA = s_prim_pairs[warpId][ij].PA.z;

        double xP = s_prim_pairs[warpId][ij].P.x;
        double yP = s_prim_pairs[warpId][ij].P.y;
        double zP = s_prim_pairs[warpId][ij].P.z;

        double eval = s_prim_pairs[warpId][ij].K_coeff_prod;
#else
        double RHO = prim_pairs[ij].gamma;
        double X_PA = prim_pairs[ij].PA.x;
        double Y_PA = prim_pairs[ij].PA.y;
        double Z_PA = prim_pairs[ij].PA.z;

        double xP = prim_pairs[ij].P.x;
        double yP = prim_pairs[ij].P.y;
        double zP = prim_pairs[ij].P.z;

        double eval = prim_pairs[ij].K_coeff_prod;
#endif

        // Evaluate T Values
        SCALAR_TYPE X_PC = SCALAR_SUB(xP, point_x);
        SCALAR_TYPE Y_PC = SCALAR_SUB(yP, point_y);
        SCALAR_TYPE Z_PC = SCALAR_SUB(zP, point_z);

        SCALAR_TYPE TVAL = SCALAR_MUL(X_PC, X_PC);
        TVAL = SCALAR_FMA(Y_PC, Y_PC, TVAL);
        TVAL = SCALAR_FMA(Z_PC, Z_PC, TVAL);
        TVAL = SCALAR_MUL(RHO, TVAL);
        
        SCALAR_TYPE t00, t01, TVAL_inv_e;

        // Evaluate Boys function
        boys_element<1>(&TVAL, &TVAL_inv_e, &t01, boys_table);

        // Evaluate VRR Buffer
        SCALAR_TYPE t10, tx;

        t00 = SCALAR_MUL(SCALAR_ADD(SCALAR_MUL(TVAL, t01), TVAL_inv_e), SCALAR_SET1(2.00000000000000000000));

        t00 = SCALAR_MUL(eval, t00);
        t01 = SCALAR_MUL(eval, t01);
        t10 = SCALAR_MUL(X_PA, t00);
        t10 = SCALAR_FNMA(X_PC, t01, t10);
        tx = temp_0;
        tx = SCALAR_ADD(tx, t10);
        temp_0 = tx;
        t10 = SCALAR_MUL(Y_PA, t00);
        t10 = SCALAR_FNMA(Y_PC, t01, t10);
        tx = temp_1;
        tx = SCALAR_ADD(tx, t10);
        temp_1 = tx;
        t10 = SCALAR_MUL(Z_PA, t00);
        t10 = SCALAR_FNMA(Z_PC, t01, t10);
        tx = temp_2;
        tx = SCALAR_ADD(tx, t10);
        temp_2 = tx;
      }

  
      if (abs(temp_0) > 1e-12 || abs(temp_1) > 1e-12 || abs(temp_2) > 1e-12) {
        const double * __restrict__ Xik = (Xi + pointIndex);
        const double * __restrict__ Xjk = (Xj + pointIndex);
        double * __restrict__ Gik = (Gi + pointIndex);
        double * __restrict__ Gjk = (Gj + pointIndex);

        SCALAR_TYPE const_value_v = weight;

        double const_value, X_ABp, Y_ABp, Z_ABp, comb_m_i, comb_n_j, comb_p_k;
        SCALAR_TYPE const_value_w;
        SCALAR_TYPE tx, ty, tz, tw, t0, t1, t2;

        X_ABp = 1.0; comb_m_i = 1.0;
        Y_ABp = 1.0; comb_n_j = 1.0;
        Z_ABp = 1.0; comb_p_k = 1.0;
        const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
        const_value_w = SCALAR_MUL(const_value_v, const_value);

        tx = SCALAR_LOAD((Xik + 0 * ldX));
        ty = SCALAR_LOAD((Xjk + 0 * ldX));

        t0 = SCALAR_MUL(temp_0, const_value_w);
        tz = SCALAR_MUL(ty, t0);
        tw = SCALAR_MUL(tx, t0);
        atomicAdd((Gik + 0 * ldG), tz);
                                       
        tx = SCALAR_LOAD((Xik + 1 * ldX));
        t1 = SCALAR_MUL(temp_1, const_value_w);
        tz = SCALAR_MUL(ty, t1);
        tw = SCALAR_FMA(tx, t1, tw);
        atomicAdd((Gik + 1 * ldG), tz);

        tx = SCALAR_LOAD((Xik + 2 * ldX));
        t2 = SCALAR_MUL(temp_2, const_value_w);
        tz = SCALAR_MUL(ty, t2);
        tw = SCALAR_FMA(tx, t2, tw);
        atomicAdd((Gik + 2 * ldG), tz);

        atomicAdd((Gjk + 0 * ldG), tw);
      }
    }
  }
  __syncwarp();
}


template <bool swap>
__global__ void 
__launch_bounds__(K10_NUMTHREADS, 1)
dev_integral_1_0_task_batched(
  int ntask, int nsubtask,
  GauXC::XCDeviceTask*                device_tasks,
  const GauXC::TaskToShellPairDevice* task2sp,
  const int4* subtasks,
  const int32_t* nprim_pairs_device,
  shell_pair** sp_ptr_device,
  double *boys_table) {

  __shared__ double4 s_task_data[K10_NUMTHREADS];

  const int warpId = threadIdx.x / K10_WARPSIZE;
  
  const int i_subtask = blockIdx.x;
  const int i_task = subtasks[i_subtask].x;
  const int point_start = subtasks[i_subtask].y;
  const int point_end = subtasks[i_subtask].z;
  const int point_count = point_end - point_start;

  const auto* task = device_tasks + i_task;

  const int npts = task->npts;

  const auto* points_x = task->points_x;
  const auto* points_y = task->points_y;
  const auto* points_z = task->points_z;
  const auto* weights = task->weights;

  const auto nsp = task2sp[i_task].nsp;


  const int npts_block = (point_count + blockDim.x - 1) / blockDim.x;

  for (int i_block = 0; i_block < npts_block; i_block++) {
    const int i = point_start + i_block * blockDim.x;

    // load point into registers
    const double point_x = points_x[i + threadIdx.x];
    const double point_y = points_y[i + threadIdx.x];
    const double point_z = points_z[i + threadIdx.x];
    const double weight = weights[i + threadIdx.x];

    s_task_data[threadIdx.x].x = point_x;
    s_task_data[threadIdx.x].y = point_y;
    s_task_data[threadIdx.x].z = point_z;
    s_task_data[threadIdx.x].w = weight;
    __syncthreads();

    for (int j = K10_NUMWARPS*blockIdx.y+warpId; j < nsp; j+=K10_NUMWARPS*gridDim.y) {
      const auto i_off = swap ? task2sp[i_task].task_shell_off_col_device[j] :
                                task2sp[i_task].task_shell_off_row_device[j];
      const auto j_off = swap ? task2sp[i_task].task_shell_off_row_device[j] :
                                task2sp[i_task].task_shell_off_col_device[j];


      const auto index =  task2sp[i_task].shell_pair_linear_idx_device[j];
      const auto* sp = sp_ptr_device[index];
      const auto nprim_pairs = nprim_pairs_device[index];

      dev_integral_1_0_task(
        i, point_count, nprim_pairs,
        s_task_data,
        sp,
        task->fmat + i_off + i,
        task->fmat + j_off + i,
        npts,
        task->gmat + i_off + i,
        task->gmat + j_off + i,
        npts,
        boys_table);
    }
    __syncthreads();
  }
}

  void integral_1_0_task_batched(
    bool swap,
    size_t ntasks, size_t nsubtask,
    int max_primpairs, size_t max_nsp,
    GauXC::XCDeviceTask*                device_tasks,
    const GauXC::TaskToShellPairDevice* task2sp,
    const std::array<int32_t, 4>*  subtasks,
    const int32_t* nprim_pairs_device,
    shell_pair** sp_ptr_device,
    double* sp_X_AB_device,
    double* sp_Y_AB_device,
    double* sp_Z_AB_device,
    double *boys_table,
    hipStream_t stream) {

    size_t xy_max = (1ul << 16) - 1;
    int nthreads = K10_NUMTHREADS;
    int nblocks_x = nsubtask;
    int nblocks_y = 8; //std::min(max_nsp,  xy_max);
    int nblocks_z = 1;
    dim3 nblocks(nblocks_x, nblocks_y, nblocks_z);

    if (swap) {
      dev_integral_1_0_task_batched<true><<<nblocks, nthreads, 0, stream>>>(
        ntasks, nsubtask, 
        device_tasks, task2sp, 
        (int4*) subtasks, nprim_pairs_device, sp_ptr_device,
        boys_table );
    } else {
      dev_integral_1_0_task_batched<false><<<nblocks, nthreads, 0, stream>>>(
        ntasks, nsubtask, 
        device_tasks, task2sp, 
        (int4*) subtasks, nprim_pairs_device, sp_ptr_device,
        boys_table );
    }
  }
}
