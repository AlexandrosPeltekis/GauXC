#include "hip/hip_runtime.h"
#include <math.h>
#include "../include/gpu/chebyshev_boys_computation.hpp"
#include "../include/gpu/integral_data_types.hpp"
#include "config_obara_saika.hpp"
#include "integral_1_0.hu"

#include <gauxc/util/div_ceil.hpp>
#include "device/xc_device_task.hpp"
#include "../../../common/shell_pair_to_task.hpp"

#define PI 3.14159265358979323846

#define MIN(a,b)			\
  ({ __typeof__ (a) _a = (a);	        \
  __typeof__ (b) _b = (b);		\
  _a < _b ? _a : _b; })

namespace XGPU {
  __inline__ __device__ void dev_integral_1_0_driver(size_t npts,
				   double *_points_x,
				   double *_points_y,
				   double *_points_z,
           shell_pair* sp,
				   double *Xi,
				   double *Xj,
				   int ldX,
				   double *Gi,
				   double *Gj,
				   int ldG, 
				   double *weights, 
				   double *boys_table) {
    //__shared__ double temp[128 * 3];
    double temp_0, temp_1, temp_2;
    const auto nprim_pairs = sp->nprim_pairs();
    const auto prim_pairs  = sp->prim_pairs();
    
    for(size_t p_outer = blockIdx.x * blockDim.x; p_outer < npts; p_outer += gridDim.x * blockDim.x) {
      double *_point_outer_x = (_points_x + p_outer);
      double *_point_outer_y = (_points_y + p_outer);
      double *_point_outer_z = (_points_z + p_outer);

      size_t p_inner = (threadIdx.x < (npts - p_outer)) ? threadIdx.x : (npts - p_outer);

      //for(int i = 0; i < 3; ++i) SCALAR_STORE((temp + i * blockDim.x + threadIdx.x), SCALAR_ZERO());
      temp_0 = SCALAR_ZERO();
      temp_1 = SCALAR_ZERO();
      temp_2 = SCALAR_ZERO();

      for(int ij = 0; ij < nprim_pairs; ++ij) {
	double RHO = prim_pairs[ij].gamma;
	double X_PA = prim_pairs[ij].PA.x;
	double Y_PA = prim_pairs[ij].PA.y;
	double Z_PA = prim_pairs[ij].PA.z;

	double xP = prim_pairs[ij].P.x;
	double yP = prim_pairs[ij].P.y;
	double zP = prim_pairs[ij].P.z;

	double eval = prim_pairs[ij].K_coeff_prod;

	// Evaluate T Values
	SCALAR_TYPE xC = SCALAR_LOAD((_point_outer_x + p_inner));
	SCALAR_TYPE yC = SCALAR_LOAD((_point_outer_y + p_inner));
	SCALAR_TYPE zC = SCALAR_LOAD((_point_outer_z + p_inner));

	SCALAR_TYPE X_PC = SCALAR_SUB(xP, xC);
	SCALAR_TYPE Y_PC = SCALAR_SUB(yP, yC);
	SCALAR_TYPE Z_PC = SCALAR_SUB(zP, zC);

	SCALAR_TYPE TVAL = SCALAR_MUL(X_PC, X_PC);
	TVAL = SCALAR_FMA(Y_PC, Y_PC, TVAL);
	TVAL = SCALAR_FMA(Z_PC, Z_PC, TVAL);
	TVAL = SCALAR_MUL(RHO, TVAL);
	
	SCALAR_TYPE t00, t01, TVAL_inv_e;

	// Evaluate Boys function
	boys_element<1>(&TVAL, &TVAL_inv_e, &t01, boys_table);

	// Evaluate VRR Buffer
	SCALAR_TYPE t10, tx;

	t00 = SCALAR_MUL(SCALAR_ADD(SCALAR_MUL(TVAL, t01), TVAL_inv_e), SCALAR_SET1(2.00000000000000000000));

	t00 = SCALAR_MUL(eval, t00);
	t01 = SCALAR_MUL(eval, t01);
	t10 = SCALAR_MUL(X_PA, t00);
	t10 = SCALAR_FNMA(X_PC, t01, t10);
	//tx = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	tx = temp_0;
	tx = SCALAR_ADD(tx, t10);
	//SCALAR_STORE((temp + 0 * blockDim.x + threadIdx.x), tx);
  temp_0 = tx;
	t10 = SCALAR_MUL(Y_PA, t00);
	t10 = SCALAR_FNMA(Y_PC, t01, t10);
	//tx = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	tx = temp_1;
	tx = SCALAR_ADD(tx, t10);
	//SCALAR_STORE((temp + 1 * blockDim.x + threadIdx.x), tx);
  temp_1 = tx;
	t10 = SCALAR_MUL(Z_PA, t00);
	t10 = SCALAR_FNMA(Z_PC, t01, t10);
	//tx = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	tx = temp_2;
	tx = SCALAR_ADD(tx, t10);
	//SCALAR_STORE((temp + 2 * blockDim.x + threadIdx.x), tx);
  temp_2 = tx;
      }

      if(threadIdx.x < npts - p_outer) {
	double *Xik = (Xi + p_outer + p_inner);
	double *Xjk = (Xj + p_outer + p_inner);
	double *Gik = (Gi + p_outer + p_inner);
	double *Gjk = (Gj + p_outer + p_inner);

	SCALAR_TYPE const_value_v = SCALAR_LOAD((weights + p_outer + p_inner));

	double const_value, X_ABp, Y_ABp, Z_ABp, comb_m_i, comb_n_j, comb_p_k;
	SCALAR_TYPE const_value_w;
	SCALAR_TYPE tx, ty, tz, tw, t0, t1, t2;

	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	//t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
  t0 = temp_0;
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	SCALAR_STORE((Gik + 0 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	//t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
  t1 = temp_1;
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	SCALAR_STORE((Gik + 1 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	//t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
  t2 = temp_2;
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	SCALAR_STORE((Gik + 2 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
      }
    }
  }

  __global__ void dev_integral_1_0(size_t npts,
				   double *points_x,
				   double *points_y,
				   double *points_z,
           shell_pair* sp,
				   double *Xi,
				   double *Xj,
				   int ldX,
				   double *Gi,
				   double *Gj,
				   int ldG, 
				   double *weights, 
				   double *boys_table) {
    dev_integral_1_0_driver( npts, points_x, points_y, points_z, sp, Xi, Xj, ldX,
      Gi, Gj, ldG, weights, boys_table );
  }

    void integral_1_0(size_t npts,
		    double *points_x,
		    double *points_y,
		    double *points_z,
        shell_pair* sp,
		    double *Xi,
		    double *Xj,
		    int ldX,
		    double *Gi,
		    double *Gj,
		    int ldG, 
		    double *weights, 
		  double *boys_table,
      hipStream_t stream) {
      dev_integral_1_0<<<320, 128, 0, stream>>>(npts,
				     points_x,
				     points_y,
				     points_z,
             sp,
				     Xi,
				     Xj,
				     ldX,
				     Gi,
				     Gj,
				     ldG, 
				     weights,
				     boys_table);
    }




  template <bool swap>
  __global__ void dev_integral_1_0_batched(
           const GauXC::ShellPairToTaskDevice* sp2task,
           GauXC::XCDeviceTask*                device_tasks,
				   double *boys_table) {

    const int ntask = sp2task->ntask;
    for( int i_task = blockIdx.y; i_task < ntask; i_task += gridDim.y ) {
    
      const auto iT = sp2task->task_idx_device[i_task];
      const auto* task  = device_tasks + iT;
      const auto  npts  = task->npts;

      int i_off, j_off;
      if constexpr ( swap ) {
        j_off = sp2task->task_shell_off_row_device[i_task]*npts;
        i_off = sp2task->task_shell_off_col_device[i_task]*npts;
      } else {
        i_off = sp2task->task_shell_off_row_device[i_task]*npts;
        j_off = sp2task->task_shell_off_col_device[i_task]*npts;
      }


      dev_integral_1_0_driver( 
        npts,
        task->points_x,
        task->points_y,
        task->points_z,
        sp2task->shell_pair_device,
        task->fmat + i_off,
        task->fmat + j_off,
        npts,
        task->gmat + i_off,
        task->gmat + j_off,
        npts,
        task->weights, boys_table );
    }

  }



  void integral_1_0_batched(bool swap, size_t ntask_sp,
        const GauXC::ShellPairToTaskDevice* sp2task,
        GauXC::XCDeviceTask*                device_tasks,
		    double *boys_table,
        hipStream_t stream) {

    int nthreads = 128;
    int nblocks_x = 160;
    int nblocks_y = ntask_sp;
    dim3 nblocks(nblocks_x, nblocks_y);

    if(swap)
      dev_integral_1_0_batched<true><<<nblocks,nthreads,0,stream>>>(
        sp2task, device_tasks, boys_table );
    else
      dev_integral_1_0_batched<false><<<nblocks,nthreads,0,stream>>>(
        sp2task, device_tasks, boys_table );

  }
}
