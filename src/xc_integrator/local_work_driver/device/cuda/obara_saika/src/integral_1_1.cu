#include "hip/hip_runtime.h"
#include <math.h>
#include "../include/gpu/chebyshev_boys_computation.hpp"
#include "config_obara_saika.hpp"
#include "integral_1_1.hu"

namespace XGPU {
  __inline__ __device__ void dev_integral_1_1_driver(double X_AB,
				   double Y_AB,
				   double Z_AB,
				   size_t npts,
				   double *_points_x,
				   double *_points_y,
				   double *_points_z,
           shell_pair* sp,
				   double *Xi,
				   double *Xj,
				   int ldX,
				   double *Gi,
				   double *Gj,
				   int ldG, 
				   double *weights, 
				   double *boys_table) {
    //__shared__ double temp[128 * 9];
    double temp_0, temp_1, temp_2, temp_3, temp_4, temp_5, temp_6, temp_7, temp_8;
    const auto nprim_pairs = sp->nprim_pairs();
    const auto prim_pairs  = sp->prim_pairs();
    
    for(size_t p_outer = blockIdx.x * blockDim.x; p_outer < npts; p_outer += gridDim.x * blockDim.x) {
      double *_point_outer_x = (_points_x + p_outer);
      double *_point_outer_y = (_points_y + p_outer);
      double *_point_outer_z = (_points_z + p_outer);

      size_t p_inner = (threadIdx.x < (npts - p_outer)) ? threadIdx.x : (npts - p_outer);

      //for(int i = 0; i < 9; ++i) SCALAR_STORE((temp + i * blockDim.x + threadIdx.x), SCALAR_ZERO());
      temp_0 = SCALAR_ZERO();
      temp_1 = SCALAR_ZERO();
      temp_2 = SCALAR_ZERO();
      temp_3 = SCALAR_ZERO();
      temp_4 = SCALAR_ZERO();
      temp_5 = SCALAR_ZERO();
      temp_6 = SCALAR_ZERO();
      temp_7 = SCALAR_ZERO();
      temp_8 = SCALAR_ZERO();


      for(int ij = 0; ij < nprim_pairs; ++ij) {
	double RHO = prim_pairs[ij].gamma;
	double RHO_INV = prim_pairs[ij].gamma_inv;
	double X_PA = prim_pairs[ij].PA.x;
	double Y_PA = prim_pairs[ij].PA.y;
	double Z_PA = prim_pairs[ij].PA.z;

	double xP = prim_pairs[ij].P.x;
	double yP = prim_pairs[ij].P.y;
	double zP = prim_pairs[ij].P.z;

	double eval = prim_pairs[ij].K_coeff_prod;
    if( std::abs(eval) < 1e-12 ) continue;

	// Evaluate T Values
	SCALAR_TYPE xC = SCALAR_LOAD((_point_outer_x + p_inner));
	SCALAR_TYPE yC = SCALAR_LOAD((_point_outer_y + p_inner));
	SCALAR_TYPE zC = SCALAR_LOAD((_point_outer_z + p_inner));

	SCALAR_TYPE X_PC = SCALAR_SUB(xP, xC);
	SCALAR_TYPE Y_PC = SCALAR_SUB(yP, yC);
	SCALAR_TYPE Z_PC = SCALAR_SUB(zP, zC);

	SCALAR_TYPE TVAL = SCALAR_MUL(X_PC, X_PC);
	TVAL = SCALAR_FMA(Y_PC, Y_PC, TVAL);
	TVAL = SCALAR_FMA(Z_PC, Z_PC, TVAL);
	TVAL = SCALAR_MUL(RHO, TVAL);

	SCALAR_TYPE t00, t01, t02, TVAL_inv_e;

	// Evaluate Boys function
	boys_element<2>(&TVAL, &TVAL_inv_e, &t02, boys_table);

	// Evaluate VRR Buffer
	SCALAR_TYPE t10, t11, t20, tx, ty;

	t01 = SCALAR_MUL(SCALAR_ADD(SCALAR_MUL(TVAL, t02), TVAL_inv_e), SCALAR_SET1(0.66666666666666662966));
	t00 = SCALAR_MUL(SCALAR_ADD(SCALAR_MUL(TVAL, t01), TVAL_inv_e), SCALAR_SET1(2.00000000000000000000));

	t00 = SCALAR_MUL(eval, t00);
	t01 = SCALAR_MUL(eval, t01);
	t02 = SCALAR_MUL(eval, t02);
	t10 = SCALAR_MUL(X_PA, t00);
	t10 = SCALAR_FNMA(X_PC, t01, t10);
	t11 = SCALAR_MUL(X_PA, t01);
	t11 = SCALAR_FNMA(X_PC, t02, t11);
	//tx = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
  tx = temp_0;
	tx = SCALAR_ADD(tx, t10);
	//SCALAR_STORE((temp + 0 * blockDim.x + threadIdx.x), tx);
  temp_0 = tx;
	t20 = SCALAR_MUL(X_PA, t10);
	t20 = SCALAR_FNMA(X_PC, t11, t20);
	tx = SCALAR_SUB(t00, t01);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t20 = SCALAR_FMA(tx, ty, t20);
	//tx = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
  tx = temp_3;
	tx = SCALAR_ADD(tx, t20);
	//SCALAR_STORE((temp + 3 * blockDim.x + threadIdx.x), tx);
  temp_3 = tx;
	t20 = SCALAR_MUL(Y_PA, t10);
	t20 = SCALAR_FNMA(Y_PC, t11, t20);
	//tx = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
  tx = temp_4;
	tx = SCALAR_ADD(tx, t20);
	//SCALAR_STORE((temp + 4 * blockDim.x + threadIdx.x), tx);
  temp_4 = tx;
	t20 = SCALAR_MUL(Z_PA, t10);
	t20 = SCALAR_FNMA(Z_PC, t11, t20);
	//tx = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
  tx = temp_5;
	tx = SCALAR_ADD(tx, t20);
	//SCALAR_STORE((temp + 5 * blockDim.x + threadIdx.x), tx);
  temp_5 = tx;
	t10 = SCALAR_MUL(Y_PA, t00);
	t10 = SCALAR_FNMA(Y_PC, t01, t10);
	t11 = SCALAR_MUL(Y_PA, t01);
	t11 = SCALAR_FNMA(Y_PC, t02, t11);
	//tx = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
  tx = temp_1;
	tx = SCALAR_ADD(tx, t10);
	//SCALAR_STORE((temp + 1 * blockDim.x + threadIdx.x), tx);
  temp_1 = tx;
	t20 = SCALAR_MUL(Y_PA, t10);
	t20 = SCALAR_FNMA(Y_PC, t11, t20);
	tx = SCALAR_SUB(t00, t01);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t20 = SCALAR_FMA(tx, ty, t20);
	//tx = SCALAR_LOAD((temp + 6 * blockDim.x + threadIdx.x));
  tx = temp_6;
	tx = SCALAR_ADD(tx, t20);
	//SCALAR_STORE((temp + 6 * blockDim.x + threadIdx.x), tx);
  temp_6 = tx;
	t20 = SCALAR_MUL(Z_PA, t10);
	t20 = SCALAR_FNMA(Z_PC, t11, t20);
	//tx = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
  tx = temp_7;
	tx = SCALAR_ADD(tx, t20);
	//SCALAR_STORE((temp + 7 * blockDim.x + threadIdx.x), tx);
  temp_7 = tx;
	t10 = SCALAR_MUL(Z_PA, t00);
	t10 = SCALAR_FNMA(Z_PC, t01, t10);
	t11 = SCALAR_MUL(Z_PA, t01);
	t11 = SCALAR_FNMA(Z_PC, t02, t11);
	//tx = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
  tx = temp_2;
	tx = SCALAR_ADD(tx, t10);
	//SCALAR_STORE((temp + 2 * blockDim.x + threadIdx.x), tx);
  temp_2 = tx;
	t20 = SCALAR_MUL(Z_PA, t10);
	t20 = SCALAR_FNMA(Z_PC, t11, t20);
	tx = SCALAR_SUB(t00, t01);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t20 = SCALAR_FMA(tx, ty, t20);
	//tx = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
  tx = temp_8;
	tx = SCALAR_ADD(tx, t20);
	//SCALAR_STORE((temp + 8 * blockDim.x + threadIdx.x), tx);
  temp_8 = tx;
      }

      if(threadIdx.x < npts - p_outer) {
	double *Xik = (Xi + p_outer + p_inner);
	double *Xjk = (Xj + p_outer + p_inner);
	double *Gik = (Gi + p_outer + p_inner);
	double *Gjk = (Gj + p_outer + p_inner);

	SCALAR_TYPE const_value_v = SCALAR_LOAD((weights + p_outer + p_inner));

	double const_value, X_ABp, Y_ABp, Z_ABp, comb_m_i, comb_n_j, comb_p_k;
	SCALAR_TYPE const_value_w;
	SCALAR_TYPE tx, ty, tz, tw, t0, t1, t2;

  #if 0
	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	//t0 = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
  t0 = temp_3;
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	SCALAR_STORE((Gik + 0 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	//t1 = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
  t1 = temp_4;
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	SCALAR_STORE((Gik + 1 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	//t2 = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
  t2 = temp_5;
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	SCALAR_STORE((Gik + 2 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	X_ABp = SCALAR_MUL(X_ABp, X_AB); comb_m_i = SCALAR_MUL(comb_m_i * 1, SCALAR_RECIPROCAL(1));
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	//t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
  t0 = temp_0;
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	SCALAR_STORE((Gik + 0 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	//t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
  t1 = temp_1;
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	SCALAR_STORE((Gik + 1 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	//t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
  t2 = temp_2;
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	SCALAR_STORE((Gik + 2 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	//t0 = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
  t0 = temp_4;
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	SCALAR_STORE((Gik + 0 * ldG), tz);
	SCALAR_STORE((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	//t1 = SCALAR_LOAD((temp + 6 * blockDim.x + threadIdx.x));
  t1 = temp_6;
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	SCALAR_STORE((Gik + 1 * ldG), tz);
	SCALAR_STORE((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	//t2 = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
  t2 = temp_7;
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	SCALAR_STORE((Gik + 2 * ldG), tz);
	SCALAR_STORE((Gjk + 1 * ldG), tw);
	Y_ABp = SCALAR_MUL(Y_ABp, Y_AB); comb_n_j = SCALAR_MUL(comb_n_j * 1, SCALAR_RECIPROCAL(1));
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	//t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
  t0 = temp_0;
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	SCALAR_STORE((Gik + 0 * ldG), tz);
	SCALAR_STORE((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	//t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
  t1 = temp_1;
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	SCALAR_STORE((Gik + 1 * ldG), tz);
	SCALAR_STORE((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	//t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
  t2 = temp_2;
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	SCALAR_STORE((Gik + 2 * ldG), tz);
	SCALAR_STORE((Gjk + 1 * ldG), tw);
	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	//t0 = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
  t0 = temp_5;
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	SCALAR_STORE((Gik + 0 * ldG), tz);
	SCALAR_STORE((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	//t1 = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
  t1 = temp_7;
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	SCALAR_STORE((Gik + 1 * ldG), tz);
	SCALAR_STORE((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	//t2 = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
  t2 = temp_8;
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	SCALAR_STORE((Gik + 2 * ldG), tz);
	SCALAR_STORE((Gjk + 2 * ldG), tw);
	Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 1, SCALAR_RECIPROCAL(1));
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	//t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
  t0 = temp_0;
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	SCALAR_STORE((Gik + 0 * ldG), tz);
	SCALAR_STORE((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	//t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
  t1 = temp_1;
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	SCALAR_STORE((Gik + 1 * ldG), tz);
	SCALAR_STORE((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	//t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
  t2 = temp_2;
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	SCALAR_STORE((Gik + 2 * ldG), tz);
	SCALAR_STORE((Gjk + 2 * ldG), tw);
  #else

  /**** j = 0 ****/
	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);

	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	t0 = SCALAR_MUL(temp_3, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_MUL(tx, t0);
	atomicAdd((Gik + 0 * ldG), tz);

	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_MUL(temp_4, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);

	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_MUL(temp_5, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);

	X_ABp = SCALAR_MUL(X_ABp, X_AB); comb_m_i = SCALAR_MUL(comb_m_i * 1, SCALAR_RECIPROCAL(1));
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);

	tx = SCALAR_LOAD((Xik + 0 * ldX));
	t0 = SCALAR_MUL(temp_0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);

	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_MUL(temp_1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);

	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_MUL(temp_2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 0 * ldG), tw);



  /**** j = 1 ****/
	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);

	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	t0 = SCALAR_MUL(temp_4, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_MUL(tx, t0);
	atomicAdd((Gik + 0 * ldG), tz);

	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_MUL(temp_6, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);

	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_MUL(temp_7, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);

	Y_ABp = SCALAR_MUL(Y_ABp, Y_AB); comb_n_j = SCALAR_MUL(comb_n_j * 1, SCALAR_RECIPROCAL(1));
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);

	tx = SCALAR_LOAD((Xik + 0 * ldX));
	t0 = SCALAR_MUL(temp_0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);

	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_MUL(temp_1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);

	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_MUL(temp_2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 1 * ldG), tw);


  /**** j = 2 ****/
	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);

	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	t0 = SCALAR_MUL(temp_5, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_MUL(tx, t0);
	atomicAdd((Gik + 0 * ldG), tz);

	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_MUL(temp_7, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);

	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_MUL(temp_8, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);

	Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 1, SCALAR_RECIPROCAL(1));
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);

	tx = SCALAR_LOAD((Xik + 0 * ldX));
	t0 = SCALAR_MUL(temp_0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_FMA(tx, t0, tw);
	atomicAdd((Gik + 0 * ldG), tz);

	tx = SCALAR_LOAD((Xik + 1 * ldX));
	t1 = SCALAR_MUL(temp_1, const_value_w);
	tz = SCALAR_MUL(ty, t1);
	tw = SCALAR_FMA(tx, t1, tw);
	atomicAdd((Gik + 1 * ldG), tz);

	tx = SCALAR_LOAD((Xik + 2 * ldX));
	t2 = SCALAR_MUL(temp_2, const_value_w);
	tz = SCALAR_MUL(ty, t2);
	tw = SCALAR_FMA(tx, t2, tw);
	atomicAdd((Gik + 2 * ldG), tz);
	atomicAdd((Gjk + 2 * ldG), tw);
  #endif
      }
    }
  }

  __global__ void dev_integral_1_1(
           double X_AB,
				   double Y_AB,
				   double Z_AB,
           size_t npts,
				   double *points_x,
				   double *points_y,
				   double *points_z,
           shell_pair* sp,
				   double *Xi,
				   double *Xj,
				   int ldX,
				   double *Gi,
				   double *Gj,
				   int ldG, 
				   double *weights, 
				   double *boys_table) {
    dev_integral_1_1_driver( X_AB, Y_AB, Z_AB, npts, points_x, points_y, 
      points_z, sp, Xi, Xj, ldX, Gi, Gj, ldG, weights, boys_table );
  }

  void integral_1_1(double X_AB,
		    double Y_AB,
		    double Z_AB,
		    size_t npts,
		    double *points_x,
		    double *points_y,
		    double *points_z,
        shell_pair* sp,
		    double *Xi,
		    double *Xj,
		    int ldX,
		    double *Gi,
		    double *Gj,
		    int ldG, 
		    double *weights, 
		  double *boys_table,
      hipStream_t stream) {
    dev_integral_1_1<<<320, 128, 0, stream>>>(X_AB,
				   Y_AB,
				   Z_AB,
				   npts,
				   points_x,
				   points_y,
				   points_z,
           sp,
				   Xi,
				   Xj,
				   ldX,
				   Gi,
				   Gj,
				   ldG, 
				   weights,
				   boys_table);
  }

  __inline__ __device__ void dev_integral_1_1_batched_driver(
           double X_AB,
				   double Y_AB,
				   double Z_AB,
           const GauXC::ShellPairToTaskDevice* sp2task,
           GauXC::XCDeviceTask*                device_tasks,
				   double *boys_table) {

    const int ntask = sp2task->ntask;
    #pragma unroll 1
    for( int i_task = blockIdx.y; i_task < ntask; i_task += gridDim.y ) {
    
      const auto iT = sp2task->task_idx_device[i_task];
      const auto* task  = device_tasks + iT;
      const auto  npts  = task->npts;

      const auto  i_off = sp2task->task_shell_off_row_device[i_task]*npts;
      const auto  j_off = sp2task->task_shell_off_col_device[i_task]*npts;


      dev_integral_1_1_driver( 
        X_AB, Y_AB, Z_AB,
        npts,
        task->points_x,
        task->points_y,
        task->points_z,
        sp2task->shell_pair_device,
        task->fmat + i_off,
        task->fmat + j_off,
        npts,
        task->gmat + i_off,
        task->gmat + j_off,
        npts,
        task->weights, boys_table );
    }

  }
  __global__ void dev_integral_1_1_batched(
           double X_AB,
				   double Y_AB,
				   double Z_AB,
           const GauXC::ShellPairToTaskDevice* sp2task,
           GauXC::XCDeviceTask*                device_tasks,
				   double *boys_table) {
   dev_integral_1_1_batched_driver(X_AB,Y_AB,Z_AB,sp2task,device_tasks,boys_table);
 }



  void integral_1_1_batched(size_t ntask_sp,
        double X_AB,
				double Y_AB,
				double Z_AB,
        const GauXC::ShellPairToTaskDevice* sp2task,
        GauXC::XCDeviceTask*                device_tasks,
		    double *boys_table,
        hipStream_t stream) {

    int nthreads = 128;
    int nblocks_x = 160;
    int nblocks_y = ntask_sp;
    dim3 nblocks(nblocks_x, nblocks_y);

    dev_integral_1_1_batched<<<nblocks,nthreads,0,stream>>>(
      X_AB, Y_AB, Z_AB, sp2task, device_tasks, boys_table );

  }



  __global__ void dev_integral_1_1_shell_batched(
           int nsp,
           const GauXC::ShellPairToTaskDevice* sp2task,
           GauXC::XCDeviceTask*                device_tasks,
				   double *boys_table) {

   for( int i = blockIdx.z; i < nsp; i += gridDim.z ) {
     auto sp = sp2task + i;
     const auto X_AB = sp->X_AB;
     const auto Y_AB = sp->Y_AB;
     const auto Z_AB = sp->Z_AB;
     dev_integral_1_1_batched_driver(X_AB,Y_AB,Z_AB,sp,device_tasks,boys_table);
   }
 }

  void integral_1_1_shell_batched(
        size_t nsp,
        size_t max_ntask,
        const GauXC::ShellPairToTaskDevice* sp2task,
        GauXC::XCDeviceTask*                device_tasks,
		    double *boys_table,
        hipStream_t stream) {

    size_t xy_max = (1ul << 16) - 1;
    int nthreads = 128;
    int nblocks_x = 1;
    int nblocks_y = std::min(max_ntask, xy_max);
    int nblocks_z = std::min(nsp,  xy_max);
    dim3 nblocks(nblocks_x, nblocks_y, nblocks_z);
    dev_integral_1_1_shell_batched<<<nblocks,nthreads,0,stream>>>(
      nsp, sp2task, device_tasks, boys_table );

  }
}
