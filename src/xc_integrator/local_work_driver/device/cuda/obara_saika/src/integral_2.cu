#include "hip/hip_runtime.h"
#include <math.h>
#include "chebyshev_boys_computation.hpp"
#include "integral_data_types.hpp"
#include "config_obara_saika.hpp"
#include "integral_2.hu"

#define PI 3.14159265358979323846

#define MIN(a,b)			\
  ({ __typeof__ (a) _a = (a);	        \
  __typeof__ (b) _b = (b);		\
  _a < _b ? _a : _b; })

__global__ void integral_2(size_t npts,
                          shell_pair *shpair,
                          double *_points,
                          double *Xi,
                          int ldX,
                          double *Gi,
                          int ldG, 
                          double *weights) {
   __shared__ double *temp;
   for(size_t p_outer = blockIdx.x * blockDim.x; p_outer < npts; p_outer += gridDim.x * blockDim.x) {
      double *_point_outer = (_points + p_outer);

      size_t p_inner = (threadIdx.x < (npts - p_outer)) ? threadIdx.x : (npts - p_outer);

      double xA = shpair[0].rA.x;
      double yA = shpair[0].rA.y;
      double zA = shpair[0].rA.z;

      for(int i = 0; i < 31; ++i) SCALAR_STORE((temp + i * blockDim.x + threadIdx.x), SCALAR_ZERO());

      for(int ij = 0; ij < shpair[0].nprim_pair; ++ij) {
         double RHO = shpair[0].prim_pairs[ij].gamma;
         double RHO_INV = 1.0 / RHO;

         constexpr double X_PA = 0.0;
         constexpr double Y_PA = 0.0;
         constexpr double Z_PA = 0.0;

         double eval = shpair[0].prim_pairs[ij].coeff_prod * shpair[0].prim_pairs[ij].K;

         // Evaluate T Values
         SCALAR_TYPE xC = SCALAR_LOAD((_point_outer + p_inner + 0 * npts));
         SCALAR_TYPE yC = SCALAR_LOAD((_point_outer + p_inner + 1 * npts));
         SCALAR_TYPE zC = SCALAR_LOAD((_point_outer + p_inner + 2 * npts));

         SCALAR_TYPE X_PC = SCALAR_SUB(xA, xC);
         SCALAR_TYPE Y_PC = SCALAR_SUB(yA, yC);
         SCALAR_TYPE Z_PC = SCALAR_SUB(zA, zC);

         X_PC = SCALAR_MUL(X_PC, X_PC);
         X_PC = SCALAR_FMA(Y_PC, Y_PC, X_PC);
         X_PC = SCALAR_FMA(Z_PC, Z_PC, X_PC);
         SCALAR_TYPE TVAL = SCALAR_MUL(RHO, X_PC);

         SCALAR_TYPE t00, t01, t02, t03, t04;

         // Evaluate Boys function
         t00 = GauXC::gauxc_boys_element<0>(TVAL);
         t01 = GauXC::gauxc_boys_element<1>(TVAL);
         t02 = GauXC::gauxc_boys_element<2>(TVAL);
         t03 = GauXC::gauxc_boys_element<3>(TVAL);
         t04 = GauXC::gauxc_boys_element<4>(TVAL);

         // Evaluate VRR Buffer
         SCALAR_TYPE t10, t11, t12, t13, t20, t21, t22, t30, t31, t40, tx, ty;

         t00 = SCALAR_MUL(eval, t00);
         t01 = SCALAR_MUL(eval, t01);
         t02 = SCALAR_MUL(eval, t02);
         t03 = SCALAR_MUL(eval, t03);
         t04 = SCALAR_MUL(eval, t04);
         t10 = SCALAR_MUL(X_PA, t00);
         t10 = SCALAR_FNMA(X_PC, t01, t10);
         t11 = SCALAR_MUL(X_PA, t01);
         t11 = SCALAR_FNMA(X_PC, t02, t11);
         t12 = SCALAR_MUL(X_PA, t02);
         t12 = SCALAR_FNMA(X_PC, t03, t12);
         t13 = SCALAR_MUL(X_PA, t03);
         t13 = SCALAR_FNMA(X_PC, t04, t13);
         t20 = SCALAR_MUL(X_PA, t10);
         t20 = SCALAR_FNMA(X_PC, t11, t20);
         tx = SCALAR_SUB(t00, t01);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t20 = SCALAR_FMA(tx, ty, t20);
         t21 = SCALAR_MUL(X_PA, t11);
         t21 = SCALAR_FNMA(X_PC, t12, t21);
         tx = SCALAR_SUB(t01, t02);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t21 = SCALAR_FMA(tx, ty, t21);
         t22 = SCALAR_MUL(X_PA, t12);
         t22 = SCALAR_FNMA(X_PC, t13, t22);
         tx = SCALAR_SUB(t02, t03);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t22 = SCALAR_FMA(tx, ty, t22);
         tx = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t20);
         SCALAR_STORE((temp + 0 * blockDim.x + threadIdx.x), tx);
         t30 = SCALAR_MUL(X_PA, t20);
         t30 = SCALAR_FNMA(X_PC, t21, t30);
         tx = SCALAR_SUB(t10, t11);
         ty = SCALAR_SET1(0.5 * 2);
         ty = SCALAR_MUL(ty, RHO_INV);
         t30 = SCALAR_FMA(tx, ty, t30);
         t31 = SCALAR_MUL(X_PA, t21);
         t31 = SCALAR_FNMA(X_PC, t22, t31);
         tx = SCALAR_SUB(t11, t12);
         ty = SCALAR_SET1(0.5 * 2);
         ty = SCALAR_MUL(ty, RHO_INV);
         t31 = SCALAR_FMA(tx, ty, t31);
         tx = SCALAR_LOAD((temp + 6 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t30);
         SCALAR_STORE((temp + 6 * blockDim.x + threadIdx.x), tx);
         t40 = SCALAR_MUL(X_PA, t30);
         t40 = SCALAR_FNMA(X_PC, t31, t40);
         tx = SCALAR_SUB(t20, t21);
         ty = SCALAR_SET1(0.5 * 3);
         ty = SCALAR_MUL(ty, RHO_INV);
         t40 = SCALAR_FMA(tx, ty, t40);
         tx = SCALAR_LOAD((temp + 16 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t40);
         SCALAR_STORE((temp + 16 * blockDim.x + threadIdx.x), tx);
         t40 = SCALAR_MUL(Y_PA, t30);
         t40 = SCALAR_FNMA(Y_PC, t31, t40);
         tx = SCALAR_LOAD((temp + 17 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t40);
         SCALAR_STORE((temp + 17 * blockDim.x + threadIdx.x), tx);
         t40 = SCALAR_MUL(Z_PA, t30);
         t40 = SCALAR_FNMA(Z_PC, t31, t40);
         tx = SCALAR_LOAD((temp + 18 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t40);
         SCALAR_STORE((temp + 18 * blockDim.x + threadIdx.x), tx);
         t30 = SCALAR_MUL(Y_PA, t20);
         t30 = SCALAR_FNMA(Y_PC, t21, t30);
         t31 = SCALAR_MUL(Y_PA, t21);
         t31 = SCALAR_FNMA(Y_PC, t22, t31);
         tx = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t30);
         SCALAR_STORE((temp + 7 * blockDim.x + threadIdx.x), tx);
         t40 = SCALAR_MUL(Y_PA, t30);
         t40 = SCALAR_FNMA(Y_PC, t31, t40);
         tx = SCALAR_SUB(t20, t21);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t40 = SCALAR_FMA(tx, ty, t40);
         tx = SCALAR_LOAD((temp + 19 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t40);
         SCALAR_STORE((temp + 19 * blockDim.x + threadIdx.x), tx);
         t40 = SCALAR_MUL(Z_PA, t30);
         t40 = SCALAR_FNMA(Z_PC, t31, t40);
         tx = SCALAR_LOAD((temp + 20 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t40);
         SCALAR_STORE((temp + 20 * blockDim.x + threadIdx.x), tx);
         t30 = SCALAR_MUL(Z_PA, t20);
         t30 = SCALAR_FNMA(Z_PC, t21, t30);
         t31 = SCALAR_MUL(Z_PA, t21);
         t31 = SCALAR_FNMA(Z_PC, t22, t31);
         tx = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t30);
         SCALAR_STORE((temp + 8 * blockDim.x + threadIdx.x), tx);
         t40 = SCALAR_MUL(Z_PA, t30);
         t40 = SCALAR_FNMA(Z_PC, t31, t40);
         tx = SCALAR_SUB(t20, t21);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t40 = SCALAR_FMA(tx, ty, t40);
         tx = SCALAR_LOAD((temp + 21 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t40);
         SCALAR_STORE((temp + 21 * blockDim.x + threadIdx.x), tx);
         t20 = SCALAR_MUL(Y_PA, t10);
         t20 = SCALAR_FNMA(Y_PC, t11, t20);
         t21 = SCALAR_MUL(Y_PA, t11);
         t21 = SCALAR_FNMA(Y_PC, t12, t21);
         t22 = SCALAR_MUL(Y_PA, t12);
         t22 = SCALAR_FNMA(Y_PC, t13, t22);
         tx = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t20);
         SCALAR_STORE((temp + 1 * blockDim.x + threadIdx.x), tx);
         t30 = SCALAR_MUL(Y_PA, t20);
         t30 = SCALAR_FNMA(Y_PC, t21, t30);
         tx = SCALAR_SUB(t10, t11);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t30 = SCALAR_FMA(tx, ty, t30);
         t31 = SCALAR_MUL(Y_PA, t21);
         t31 = SCALAR_FNMA(Y_PC, t22, t31);
         tx = SCALAR_SUB(t11, t12);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t31 = SCALAR_FMA(tx, ty, t31);
         tx = SCALAR_LOAD((temp + 9 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t30);
         SCALAR_STORE((temp + 9 * blockDim.x + threadIdx.x), tx);
         t40 = SCALAR_MUL(Y_PA, t30);
         t40 = SCALAR_FNMA(Y_PC, t31, t40);
         tx = SCALAR_SUB(t20, t21);
         ty = SCALAR_SET1(0.5 * 2);
         ty = SCALAR_MUL(ty, RHO_INV);
         t40 = SCALAR_FMA(tx, ty, t40);
         tx = SCALAR_LOAD((temp + 22 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t40);
         SCALAR_STORE((temp + 22 * blockDim.x + threadIdx.x), tx);
         t40 = SCALAR_MUL(Z_PA, t30);
         t40 = SCALAR_FNMA(Z_PC, t31, t40);
         tx = SCALAR_LOAD((temp + 23 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t40);
         SCALAR_STORE((temp + 23 * blockDim.x + threadIdx.x), tx);
         t30 = SCALAR_MUL(Z_PA, t20);
         t30 = SCALAR_FNMA(Z_PC, t21, t30);
         t31 = SCALAR_MUL(Z_PA, t21);
         t31 = SCALAR_FNMA(Z_PC, t22, t31);
         tx = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t30);
         SCALAR_STORE((temp + 10 * blockDim.x + threadIdx.x), tx);
         t40 = SCALAR_MUL(Z_PA, t30);
         t40 = SCALAR_FNMA(Z_PC, t31, t40);
         tx = SCALAR_SUB(t20, t21);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t40 = SCALAR_FMA(tx, ty, t40);
         tx = SCALAR_LOAD((temp + 24 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t40);
         SCALAR_STORE((temp + 24 * blockDim.x + threadIdx.x), tx);
         t20 = SCALAR_MUL(Z_PA, t10);
         t20 = SCALAR_FNMA(Z_PC, t11, t20);
         t21 = SCALAR_MUL(Z_PA, t11);
         t21 = SCALAR_FNMA(Z_PC, t12, t21);
         t22 = SCALAR_MUL(Z_PA, t12);
         t22 = SCALAR_FNMA(Z_PC, t13, t22);
         tx = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t20);
         SCALAR_STORE((temp + 2 * blockDim.x + threadIdx.x), tx);
         t30 = SCALAR_MUL(Z_PA, t20);
         t30 = SCALAR_FNMA(Z_PC, t21, t30);
         tx = SCALAR_SUB(t10, t11);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t30 = SCALAR_FMA(tx, ty, t30);
         t31 = SCALAR_MUL(Z_PA, t21);
         t31 = SCALAR_FNMA(Z_PC, t22, t31);
         tx = SCALAR_SUB(t11, t12);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t31 = SCALAR_FMA(tx, ty, t31);
         tx = SCALAR_LOAD((temp + 11 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t30);
         SCALAR_STORE((temp + 11 * blockDim.x + threadIdx.x), tx);
         t40 = SCALAR_MUL(Z_PA, t30);
         t40 = SCALAR_FNMA(Z_PC, t31, t40);
         tx = SCALAR_SUB(t20, t21);
         ty = SCALAR_SET1(0.5 * 2);
         ty = SCALAR_MUL(ty, RHO_INV);
         t40 = SCALAR_FMA(tx, ty, t40);
         tx = SCALAR_LOAD((temp + 25 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t40);
         SCALAR_STORE((temp + 25 * blockDim.x + threadIdx.x), tx);
         t10 = SCALAR_MUL(Y_PA, t00);
         t10 = SCALAR_FNMA(Y_PC, t01, t10);
         t11 = SCALAR_MUL(Y_PA, t01);
         t11 = SCALAR_FNMA(Y_PC, t02, t11);
         t12 = SCALAR_MUL(Y_PA, t02);
         t12 = SCALAR_FNMA(Y_PC, t03, t12);
         t13 = SCALAR_MUL(Y_PA, t03);
         t13 = SCALAR_FNMA(Y_PC, t04, t13);
         t20 = SCALAR_MUL(Y_PA, t10);
         t20 = SCALAR_FNMA(Y_PC, t11, t20);
         tx = SCALAR_SUB(t00, t01);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t20 = SCALAR_FMA(tx, ty, t20);
         t21 = SCALAR_MUL(Y_PA, t11);
         t21 = SCALAR_FNMA(Y_PC, t12, t21);
         tx = SCALAR_SUB(t01, t02);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t21 = SCALAR_FMA(tx, ty, t21);
         t22 = SCALAR_MUL(Y_PA, t12);
         t22 = SCALAR_FNMA(Y_PC, t13, t22);
         tx = SCALAR_SUB(t02, t03);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t22 = SCALAR_FMA(tx, ty, t22);
         tx = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t20);
         SCALAR_STORE((temp + 3 * blockDim.x + threadIdx.x), tx);
         t30 = SCALAR_MUL(Y_PA, t20);
         t30 = SCALAR_FNMA(Y_PC, t21, t30);
         tx = SCALAR_SUB(t10, t11);
         ty = SCALAR_SET1(0.5 * 2);
         ty = SCALAR_MUL(ty, RHO_INV);
         t30 = SCALAR_FMA(tx, ty, t30);
         t31 = SCALAR_MUL(Y_PA, t21);
         t31 = SCALAR_FNMA(Y_PC, t22, t31);
         tx = SCALAR_SUB(t11, t12);
         ty = SCALAR_SET1(0.5 * 2);
         ty = SCALAR_MUL(ty, RHO_INV);
         t31 = SCALAR_FMA(tx, ty, t31);
         tx = SCALAR_LOAD((temp + 12 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t30);
         SCALAR_STORE((temp + 12 * blockDim.x + threadIdx.x), tx);
         t40 = SCALAR_MUL(Y_PA, t30);
         t40 = SCALAR_FNMA(Y_PC, t31, t40);
         tx = SCALAR_SUB(t20, t21);
         ty = SCALAR_SET1(0.5 * 3);
         ty = SCALAR_MUL(ty, RHO_INV);
         t40 = SCALAR_FMA(tx, ty, t40);
         tx = SCALAR_LOAD((temp + 26 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t40);
         SCALAR_STORE((temp + 26 * blockDim.x + threadIdx.x), tx);
         t40 = SCALAR_MUL(Z_PA, t30);
         t40 = SCALAR_FNMA(Z_PC, t31, t40);
         tx = SCALAR_LOAD((temp + 27 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t40);
         SCALAR_STORE((temp + 27 * blockDim.x + threadIdx.x), tx);
         t30 = SCALAR_MUL(Z_PA, t20);
         t30 = SCALAR_FNMA(Z_PC, t21, t30);
         t31 = SCALAR_MUL(Z_PA, t21);
         t31 = SCALAR_FNMA(Z_PC, t22, t31);
         tx = SCALAR_LOAD((temp + 13 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t30);
         SCALAR_STORE((temp + 13 * blockDim.x + threadIdx.x), tx);
         t40 = SCALAR_MUL(Z_PA, t30);
         t40 = SCALAR_FNMA(Z_PC, t31, t40);
         tx = SCALAR_SUB(t20, t21);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t40 = SCALAR_FMA(tx, ty, t40);
         tx = SCALAR_LOAD((temp + 28 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t40);
         SCALAR_STORE((temp + 28 * blockDim.x + threadIdx.x), tx);
         t20 = SCALAR_MUL(Z_PA, t10);
         t20 = SCALAR_FNMA(Z_PC, t11, t20);
         t21 = SCALAR_MUL(Z_PA, t11);
         t21 = SCALAR_FNMA(Z_PC, t12, t21);
         t22 = SCALAR_MUL(Z_PA, t12);
         t22 = SCALAR_FNMA(Z_PC, t13, t22);
         tx = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t20);
         SCALAR_STORE((temp + 4 * blockDim.x + threadIdx.x), tx);
         t30 = SCALAR_MUL(Z_PA, t20);
         t30 = SCALAR_FNMA(Z_PC, t21, t30);
         tx = SCALAR_SUB(t10, t11);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t30 = SCALAR_FMA(tx, ty, t30);
         t31 = SCALAR_MUL(Z_PA, t21);
         t31 = SCALAR_FNMA(Z_PC, t22, t31);
         tx = SCALAR_SUB(t11, t12);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t31 = SCALAR_FMA(tx, ty, t31);
         tx = SCALAR_LOAD((temp + 14 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t30);
         SCALAR_STORE((temp + 14 * blockDim.x + threadIdx.x), tx);
         t40 = SCALAR_MUL(Z_PA, t30);
         t40 = SCALAR_FNMA(Z_PC, t31, t40);
         tx = SCALAR_SUB(t20, t21);
         ty = SCALAR_SET1(0.5 * 2);
         ty = SCALAR_MUL(ty, RHO_INV);
         t40 = SCALAR_FMA(tx, ty, t40);
         tx = SCALAR_LOAD((temp + 29 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t40);
         SCALAR_STORE((temp + 29 * blockDim.x + threadIdx.x), tx);
         t10 = SCALAR_MUL(Z_PA, t00);
         t10 = SCALAR_FNMA(Z_PC, t01, t10);
         t11 = SCALAR_MUL(Z_PA, t01);
         t11 = SCALAR_FNMA(Z_PC, t02, t11);
         t12 = SCALAR_MUL(Z_PA, t02);
         t12 = SCALAR_FNMA(Z_PC, t03, t12);
         t13 = SCALAR_MUL(Z_PA, t03);
         t13 = SCALAR_FNMA(Z_PC, t04, t13);
         t20 = SCALAR_MUL(Z_PA, t10);
         t20 = SCALAR_FNMA(Z_PC, t11, t20);
         tx = SCALAR_SUB(t00, t01);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t20 = SCALAR_FMA(tx, ty, t20);
         t21 = SCALAR_MUL(Z_PA, t11);
         t21 = SCALAR_FNMA(Z_PC, t12, t21);
         tx = SCALAR_SUB(t01, t02);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t21 = SCALAR_FMA(tx, ty, t21);
         t22 = SCALAR_MUL(Z_PA, t12);
         t22 = SCALAR_FNMA(Z_PC, t13, t22);
         tx = SCALAR_SUB(t02, t03);
         ty = SCALAR_SET1(0.5 * 1);
         ty = SCALAR_MUL(ty, RHO_INV);
         t22 = SCALAR_FMA(tx, ty, t22);
         tx = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t20);
         SCALAR_STORE((temp + 5 * blockDim.x + threadIdx.x), tx);
         t30 = SCALAR_MUL(Z_PA, t20);
         t30 = SCALAR_FNMA(Z_PC, t21, t30);
         tx = SCALAR_SUB(t10, t11);
         ty = SCALAR_SET1(0.5 * 2);
         ty = SCALAR_MUL(ty, RHO_INV);
         t30 = SCALAR_FMA(tx, ty, t30);
         t31 = SCALAR_MUL(Z_PA, t21);
         t31 = SCALAR_FNMA(Z_PC, t22, t31);
         tx = SCALAR_SUB(t11, t12);
         ty = SCALAR_SET1(0.5 * 2);
         ty = SCALAR_MUL(ty, RHO_INV);
         t31 = SCALAR_FMA(tx, ty, t31);
         tx = SCALAR_LOAD((temp + 15 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t30);
         SCALAR_STORE((temp + 15 * blockDim.x + threadIdx.x), tx);
         t40 = SCALAR_MUL(Z_PA, t30);
         t40 = SCALAR_FNMA(Z_PC, t31, t40);
         tx = SCALAR_SUB(t20, t21);
         ty = SCALAR_SET1(0.5 * 3);
         ty = SCALAR_MUL(ty, RHO_INV);
         t40 = SCALAR_FMA(tx, ty, t40);
         tx = SCALAR_LOAD((temp + 30 * blockDim.x + threadIdx.x));
         tx = SCALAR_ADD(tx, t40);
         SCALAR_STORE((temp + 30 * blockDim.x + threadIdx.x), tx);
      }

      double *Xik = (Xi + p_outer + p_inner);
      double *Gik = (Gi + p_outer + p_inner);

      SCALAR_TYPE tx, wg, xik, gik;
      tx  = SCALAR_LOAD((temp + 16 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 0 * ldX));
      gik = SCALAR_LOAD((Gik + 0 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 0 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 17 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 0 * ldX));
      gik = SCALAR_LOAD((Gik + 1 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 1 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 18 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 0 * ldX));
      gik = SCALAR_LOAD((Gik + 2 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 2 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 19 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 0 * ldX));
      gik = SCALAR_LOAD((Gik + 3 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 3 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 20 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 0 * ldX));
      gik = SCALAR_LOAD((Gik + 4 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 4 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 21 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 0 * ldX));
      gik = SCALAR_LOAD((Gik + 5 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 5 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 17 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 1 * ldX));
      gik = SCALAR_LOAD((Gik + 0 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 0 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 19 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 1 * ldX));
      gik = SCALAR_LOAD((Gik + 1 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 1 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 20 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 1 * ldX));
      gik = SCALAR_LOAD((Gik + 2 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 2 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 22 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 1 * ldX));
      gik = SCALAR_LOAD((Gik + 3 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 3 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 23 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 1 * ldX));
      gik = SCALAR_LOAD((Gik + 4 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 4 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 24 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 1 * ldX));
      gik = SCALAR_LOAD((Gik + 5 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 5 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 18 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 2 * ldX));
      gik = SCALAR_LOAD((Gik + 0 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 0 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 20 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 2 * ldX));
      gik = SCALAR_LOAD((Gik + 1 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 1 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 21 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 2 * ldX));
      gik = SCALAR_LOAD((Gik + 2 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 2 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 23 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 2 * ldX));
      gik = SCALAR_LOAD((Gik + 3 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 3 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 24 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 2 * ldX));
      gik = SCALAR_LOAD((Gik + 4 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 4 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 25 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 2 * ldX));
      gik = SCALAR_LOAD((Gik + 5 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 5 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 19 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 3 * ldX));
      gik = SCALAR_LOAD((Gik + 0 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 0 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 22 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 3 * ldX));
      gik = SCALAR_LOAD((Gik + 1 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 1 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 23 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 3 * ldX));
      gik = SCALAR_LOAD((Gik + 2 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 2 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 26 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 3 * ldX));
      gik = SCALAR_LOAD((Gik + 3 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 3 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 27 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 3 * ldX));
      gik = SCALAR_LOAD((Gik + 4 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 4 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 28 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 3 * ldX));
      gik = SCALAR_LOAD((Gik + 5 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 5 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 20 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 4 * ldX));
      gik = SCALAR_LOAD((Gik + 0 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 0 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 23 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 4 * ldX));
      gik = SCALAR_LOAD((Gik + 1 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 1 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 24 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 4 * ldX));
      gik = SCALAR_LOAD((Gik + 2 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 2 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 27 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 4 * ldX));
      gik = SCALAR_LOAD((Gik + 3 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 3 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 28 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 4 * ldX));
      gik = SCALAR_LOAD((Gik + 4 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 4 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 29 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 4 * ldX));
      gik = SCALAR_LOAD((Gik + 5 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 5 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 21 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 5 * ldX));
      gik = SCALAR_LOAD((Gik + 0 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 0 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 24 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 5 * ldX));
      gik = SCALAR_LOAD((Gik + 1 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 1 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 25 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 5 * ldX));
      gik = SCALAR_LOAD((Gik + 2 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 2 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 28 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 5 * ldX));
      gik = SCALAR_LOAD((Gik + 3 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 3 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 29 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 5 * ldX));
      gik = SCALAR_LOAD((Gik + 4 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 4 * ldG), gik);
      tx  = SCALAR_LOAD((temp + 30 * blockDim.x + threadIdx.x));
      wg  = SCALAR_LOAD((weights + p_outer + p_inner));

      xik = SCALAR_LOAD((Xik + 5 * ldX));
      gik = SCALAR_LOAD((Gik + 5 * ldG));

      tx = SCALAR_MUL(tx, wg);
      gik = SCALAR_FMA(tx, xik, gik);
      SCALAR_STORE((Gik + 5 * ldG), gik);
   }
}