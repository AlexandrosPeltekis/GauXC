#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "../include/gpu/integral_data_types.hpp"
#include "../include/gpu/obara_saika_integrals.hpp"
#include "integral_0.hu"
#include "integral_1.hu"
#include "integral_2.hu"
#include "integral_0_0.hu"
#include "integral_1_0.hu"
#include "integral_1_1.hu"
#include "integral_2_0.hu"
#include "integral_2_1.hu"
#include "integral_2_2.hu"
namespace XGPU {

  void generate_shell_pair( const shells& A, const shells& B, prim_pair *prim_pairs) {
    // L Values
    const auto xA = A.origin.x;
    const auto yA = A.origin.y;
    const auto zA = A.origin.z;

    const auto xB = B.origin.x;
    const auto yB = B.origin.y;
    const auto zB = B.origin.z;

    double rABx = xA - xB;
    double rABy = yA - yB;
    double rABz = zA - zB;

    const double dAB = rABx*rABx + rABy*rABy + rABz*rABz;

    const int nprim_A = A.m;
    const int nprim_B = B.m;

    for(int i = 0, ij = 0; i < nprim_A; ++i       )
      for(int j = 0        ; j < nprim_B; ++j, ++ij ) {
	auto& pair = prim_pairs[ij];
	const auto alpha_A = A.coeff[i].alpha;
	const auto alpha_B = B.coeff[j].alpha;

	pair.gamma = alpha_A + alpha_B;
	pair.gamma_inv = 1. / pair.gamma;

	pair.P.x = (alpha_A * xA + alpha_B * xB) * pair.gamma_inv;
	pair.P.y = (alpha_A * yA + alpha_B * yB) * pair.gamma_inv;
	pair.P.z = (alpha_A * zA + alpha_B * zB) * pair.gamma_inv;

	pair.PA.x = pair.P.x - xA;
	pair.PA.y = pair.P.y - yA;
	pair.PA.z = pair.P.z - zA;

	pair.PB.x = pair.P.x - xB;
	pair.PB.y = pair.P.y - yB;
	pair.PB.z = pair.P.z - zB;

	pair.K_coeff_prod = 2 * M_PI * A.coeff[i].coeff * B.coeff[j].coeff * pair.gamma_inv * std::exp( - alpha_A * alpha_B * dAB * pair.gamma_inv );
      }

  }

  void compute_integral_shell_pair(int is_diag,
				   size_t npts,
				   double *points,
				   int lA,
				   int lB,
				   point rA,
				   point rB,
				   int nprim_pairs,
				   prim_pair *prim_pairs,
				   double *Xi,
				   double *Xj,
				   int ldX,
				   double *Gi,
				   double *Gj,
				   int ldG, 
				   double *weights,
				   double *boys_table) {
    if (is_diag) {
      if(lA == 0) {
	integral_0<<<320, 128, 128 * 1 * sizeof(double)>>>(npts,
							   points,
							   rA,
							   rB,
							   nprim_pairs,
							   prim_pairs,
							   Xi,
							   ldX,
							   Gi,
							   ldG, 
							   weights, 
							   boys_table);
      } else if(lA == 1) {
        integral_0<<<320, 128, 128 * 9 * sizeof(double)>>>(npts,
							   points,
							   rA,
							   rB,
							   nprim_pairs,
							   prim_pairs,
							   Xi,
							   ldX,
							   Gi,
							   ldG, 
							   weights, 
							   boys_table);
      } else if(lA == 2) {
        integral_0<<<320, 128, 128 * 31 * sizeof(double)>>>(npts,
							    points,
							    rA,
							    rB,
							    nprim_pairs,
							    prim_pairs,
							    Xi,
							    ldX,
							    Gi,
							    ldG, 
							    weights, 
							    boys_table);
      } else {
	printf("Type not defined!\n");
      }
    } else {
      if((lA == 0) && (lB == 0)) {
	integral_0_0<<<320, 128, 128 * 1 * sizeof(double)>>>(npts,
							     points,
							     rA,
							     rB,
							     nprim_pairs,
							     prim_pairs,
							     Xi,
							     Xj,
							     ldX,
							     Gi,
							     Gj,
							     ldG, 
							     weights,
							     boys_table);
      } else if((lA == 1) && (lB == 0)) {
	integral_1_0<<<320, 128, 128 * 3 * sizeof(double)>>>(npts,
							     points,
							     rA,
							     rB,
							     nprim_pairs,
							     prim_pairs,
							     Xi,
							     Xj,
							     ldX,
							     Gi,
							     Gj,
							     ldG, 
							     weights,
							     boys_table);
      } else if((lA == 0) && (lB == 1)) {
	integral_1_0<<<320, 128, 128 * 3 * sizeof(double)>>>(npts,
							     points,
							     rB,
							     rA,
							     nprim_pairs,
							     prim_pairs,
							     Xj,
							     Xi,
							     ldX,
							     Gj,
							     Gi,
							     ldG, 
							     weights, 
							     boys_table);
      } else if((lA == 1) && (lB == 1)) {
        integral_1_1<<<320, 128, 128 * 9 * sizeof(double)>>>(npts,
							     points,
							     rA,
							     rB,
							     nprim_pairs,
							     prim_pairs,
							     Xi,
							     Xj,
							     ldX,
							     Gi,
							     Gj,
							     ldG, 
							     weights,
							     boys_table);
      } else if((lA == 2) && (lB == 0)) {
	integral_2_0<<<320, 128, 128 * 6 * sizeof(double)>>>(npts,
							     points,
							     rA,
							     rB,
							     nprim_pairs,
							     prim_pairs,
							     Xi,
							     Xj,
							     ldX,
							     Gi,
							     Gj,
							     ldG, 
							     weights,
							     boys_table);
      } else if((lA == 0) && (lB == 2)) {
	integral_2_0<<<320, 128, 128 * 6 * sizeof(double)>>>(npts,
							     points,
							     rB,
							     rA,
							     nprim_pairs,
							     prim_pairs,
							     Xj,
							     Xi,
							     ldX,
							     Gj,
							     Gi,
							     ldG, 
							     weights, 
							     boys_table);
      } else if((lA == 2) && (lB == 1)) {
	integral_2_1<<<320, 128, 128 * 16 * sizeof(double)>>>(npts,
							      points,
							      rA,
							      rB,
							      nprim_pairs,
							      prim_pairs,
							      Xi,
							      Xj,
							      ldX,
							      Gi,
							      Gj,
							      ldG, 
							      weights,
							      boys_table);
      } else if((lA == 1) && (lB == 2)) {
	integral_2_1<<<320, 128, 128 * 16 * sizeof(double)>>>(npts,
							      points,
							      rB,
							      rA,
							      nprim_pairs,
							      prim_pairs,
							      Xj,
							      Xi,
							      ldX,
							      Gj,
							      Gi,
							      ldG, 
							      weights, 
							      boys_table);
      } else if((lA == 2) && (lB == 2)) {
        integral_2_2<<<320, 128, 128 * 31 * sizeof(double)>>>(npts,
							      points,
							      rA,
							      rB,
							      nprim_pairs,
							      prim_pairs,
							      Xi,
							      Xj,
							      ldX,
							      Gi,
							      Gj,
							      ldG, 
							      weights,
							      boys_table);
      } else {
	printf("Type not defined!\n");
      }
    }
  }
}
