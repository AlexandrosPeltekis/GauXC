#include "hip/hip_runtime.h"
#include <math.h>
#include "../include/gpu/chebyshev_boys_computation.hpp"
#include "../include/gpu/integral_data_types.hpp"
#include "config_obara_saika.hpp"
#include "integral_2_1.hu"
 
#define PI 3.14159265358979323846

#define MIN(a,b)				\
  ({ __typeof__ (a) _a = (a);			\
    __typeof__ (b) _b = (b);			\
    _a < _b ? _a : _b; })

namespace XGPU {
  __global__ void dev_integral_2_1(double X_AB,
				   double Y_AB,
				   double Z_AB,
				   size_t npts,
				   double *_points,
           shell_pair* sp,
				   double *Xi,
				   double *Xj,
				   int ldX,
				   double *Gi,
				   double *Gj,
				   int ldG, 
				   double *weights, 
				   double *boys_table) {
    __shared__ double temp[128 * 16];
    const auto nprim_pairs = sp->nprim_pairs();
    const auto prim_pairs  = sp->prim_pairs();
    for(size_t p_outer = blockIdx.x * blockDim.x; p_outer < npts; p_outer += gridDim.x * blockDim.x) {
      double *_point_outer = (_points + p_outer);

      size_t p_inner = (threadIdx.x < (npts - p_outer)) ? threadIdx.x : (npts - p_outer);

      for(int i = 0; i < 16; ++i) SCALAR_STORE((temp + i * blockDim.x + threadIdx.x), SCALAR_ZERO());

      for(int ij = 0; ij < nprim_pairs; ++ij) {
	double RHO = prim_pairs[ij].gamma;
	double RHO_INV = prim_pairs[ij].gamma_inv;
	double X_PA = prim_pairs[ij].PA.x;
	double Y_PA = prim_pairs[ij].PA.y;
	double Z_PA = prim_pairs[ij].PA.z;

	double xP = prim_pairs[ij].P.x;
	double yP = prim_pairs[ij].P.y;
	double zP = prim_pairs[ij].P.z;

	double eval = prim_pairs[ij].K_coeff_prod;

	// Evaluate T Values
	SCALAR_TYPE xC = SCALAR_LOAD((_point_outer + p_inner + 0 * npts));
	SCALAR_TYPE yC = SCALAR_LOAD((_point_outer + p_inner + 1 * npts));
	SCALAR_TYPE zC = SCALAR_LOAD((_point_outer + p_inner + 2 * npts));

	SCALAR_TYPE X_PC = SCALAR_SUB(xP, xC);
	SCALAR_TYPE Y_PC = SCALAR_SUB(yP, yC);
	SCALAR_TYPE Z_PC = SCALAR_SUB(zP, zC);

	SCALAR_TYPE TVAL = SCALAR_MUL(X_PC, X_PC);
	TVAL = SCALAR_FMA(Y_PC, Y_PC, TVAL);
	TVAL = SCALAR_FMA(Z_PC, Z_PC, TVAL);
	TVAL = SCALAR_MUL(RHO, TVAL);

	SCALAR_TYPE t00, t01, t02, t03, TVAL_inv_e;

	// Evaluate Boys function
	boys_element<3>(&TVAL, &TVAL_inv_e, &t03, boys_table);

	// Evaluate VRR Buffer
	SCALAR_TYPE t10, t11, t12, t20, t21, t30, tx, ty;

	t02 = SCALAR_MUL(SCALAR_ADD(SCALAR_MUL(TVAL, t03), TVAL_inv_e), SCALAR_SET1(0.40000000000000002220));
	t01 = SCALAR_MUL(SCALAR_ADD(SCALAR_MUL(TVAL, t02), TVAL_inv_e), SCALAR_SET1(0.66666666666666662966));
	t00 = SCALAR_MUL(SCALAR_ADD(SCALAR_MUL(TVAL, t01), TVAL_inv_e), SCALAR_SET1(2.00000000000000000000));

	t00 = SCALAR_MUL(eval, t00);
	t01 = SCALAR_MUL(eval, t01);
	t02 = SCALAR_MUL(eval, t02);
	t03 = SCALAR_MUL(eval, t03);
	t10 = SCALAR_MUL(X_PA, t00);
	t10 = SCALAR_FNMA(X_PC, t01, t10);
	t11 = SCALAR_MUL(X_PA, t01);
	t11 = SCALAR_FNMA(X_PC, t02, t11);
	t12 = SCALAR_MUL(X_PA, t02);
	t12 = SCALAR_FNMA(X_PC, t03, t12);
	t20 = SCALAR_MUL(X_PA, t10);
	t20 = SCALAR_FNMA(X_PC, t11, t20);
	tx = SCALAR_SUB(t00, t01);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t20 = SCALAR_FMA(tx, ty, t20);
	t21 = SCALAR_MUL(X_PA, t11);
	t21 = SCALAR_FNMA(X_PC, t12, t21);
	tx = SCALAR_SUB(t01, t02);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t21 = SCALAR_FMA(tx, ty, t21);
	tx = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t20);
	SCALAR_STORE((temp + 0 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(X_PA, t20);
	t30 = SCALAR_FNMA(X_PC, t21, t30);
	tx = SCALAR_SUB(t10, t11);
	ty = SCALAR_SET1(0.5 * 2);
	ty = SCALAR_MUL(ty, RHO_INV);
	t30 = SCALAR_FMA(tx, ty, t30);
	tx = SCALAR_LOAD((temp + 6 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 6 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(Y_PA, t20);
	t30 = SCALAR_FNMA(Y_PC, t21, t30);
	tx = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 7 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(Z_PA, t20);
	t30 = SCALAR_FNMA(Z_PC, t21, t30);
	tx = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 8 * blockDim.x + threadIdx.x), tx);
	t20 = SCALAR_MUL(Y_PA, t10);
	t20 = SCALAR_FNMA(Y_PC, t11, t20);
	t21 = SCALAR_MUL(Y_PA, t11);
	t21 = SCALAR_FNMA(Y_PC, t12, t21);
	tx = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t20);
	SCALAR_STORE((temp + 1 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(Y_PA, t20);
	t30 = SCALAR_FNMA(Y_PC, t21, t30);
	tx = SCALAR_SUB(t10, t11);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t30 = SCALAR_FMA(tx, ty, t30);
	tx = SCALAR_LOAD((temp + 9 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 9 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(Z_PA, t20);
	t30 = SCALAR_FNMA(Z_PC, t21, t30);
	tx = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 10 * blockDim.x + threadIdx.x), tx);
	t20 = SCALAR_MUL(Z_PA, t10);
	t20 = SCALAR_FNMA(Z_PC, t11, t20);
	t21 = SCALAR_MUL(Z_PA, t11);
	t21 = SCALAR_FNMA(Z_PC, t12, t21);
	tx = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t20);
	SCALAR_STORE((temp + 2 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(Z_PA, t20);
	t30 = SCALAR_FNMA(Z_PC, t21, t30);
	tx = SCALAR_SUB(t10, t11);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t30 = SCALAR_FMA(tx, ty, t30);
	tx = SCALAR_LOAD((temp + 11 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 11 * blockDim.x + threadIdx.x), tx);
	t10 = SCALAR_MUL(Y_PA, t00);
	t10 = SCALAR_FNMA(Y_PC, t01, t10);
	t11 = SCALAR_MUL(Y_PA, t01);
	t11 = SCALAR_FNMA(Y_PC, t02, t11);
	t12 = SCALAR_MUL(Y_PA, t02);
	t12 = SCALAR_FNMA(Y_PC, t03, t12);
	t20 = SCALAR_MUL(Y_PA, t10);
	t20 = SCALAR_FNMA(Y_PC, t11, t20);
	tx = SCALAR_SUB(t00, t01);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t20 = SCALAR_FMA(tx, ty, t20);
	t21 = SCALAR_MUL(Y_PA, t11);
	t21 = SCALAR_FNMA(Y_PC, t12, t21);
	tx = SCALAR_SUB(t01, t02);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t21 = SCALAR_FMA(tx, ty, t21);
	tx = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t20);
	SCALAR_STORE((temp + 3 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(Y_PA, t20);
	t30 = SCALAR_FNMA(Y_PC, t21, t30);
	tx = SCALAR_SUB(t10, t11);
	ty = SCALAR_SET1(0.5 * 2);
	ty = SCALAR_MUL(ty, RHO_INV);
	t30 = SCALAR_FMA(tx, ty, t30);
	tx = SCALAR_LOAD((temp + 12 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 12 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(Z_PA, t20);
	t30 = SCALAR_FNMA(Z_PC, t21, t30);
	tx = SCALAR_LOAD((temp + 13 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 13 * blockDim.x + threadIdx.x), tx);
	t20 = SCALAR_MUL(Z_PA, t10);
	t20 = SCALAR_FNMA(Z_PC, t11, t20);
	t21 = SCALAR_MUL(Z_PA, t11);
	t21 = SCALAR_FNMA(Z_PC, t12, t21);
	tx = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t20);
	SCALAR_STORE((temp + 4 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(Z_PA, t20);
	t30 = SCALAR_FNMA(Z_PC, t21, t30);
	tx = SCALAR_SUB(t10, t11);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t30 = SCALAR_FMA(tx, ty, t30);
	tx = SCALAR_LOAD((temp + 14 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 14 * blockDim.x + threadIdx.x), tx);
	t10 = SCALAR_MUL(Z_PA, t00);
	t10 = SCALAR_FNMA(Z_PC, t01, t10);
	t11 = SCALAR_MUL(Z_PA, t01);
	t11 = SCALAR_FNMA(Z_PC, t02, t11);
	t12 = SCALAR_MUL(Z_PA, t02);
	t12 = SCALAR_FNMA(Z_PC, t03, t12);
	t20 = SCALAR_MUL(Z_PA, t10);
	t20 = SCALAR_FNMA(Z_PC, t11, t20);
	tx = SCALAR_SUB(t00, t01);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t20 = SCALAR_FMA(tx, ty, t20);
	t21 = SCALAR_MUL(Z_PA, t11);
	t21 = SCALAR_FNMA(Z_PC, t12, t21);
	tx = SCALAR_SUB(t01, t02);
	ty = SCALAR_SET1(0.5 * 1);
	ty = SCALAR_MUL(ty, RHO_INV);
	t21 = SCALAR_FMA(tx, ty, t21);
	tx = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t20);
	SCALAR_STORE((temp + 5 * blockDim.x + threadIdx.x), tx);
	t30 = SCALAR_MUL(Z_PA, t20);
	t30 = SCALAR_FNMA(Z_PC, t21, t30);
	tx = SCALAR_SUB(t10, t11);
	ty = SCALAR_SET1(0.5 * 2);
	ty = SCALAR_MUL(ty, RHO_INV);
	t30 = SCALAR_FMA(tx, ty, t30);
	tx = SCALAR_LOAD((temp + 15 * blockDim.x + threadIdx.x));
	tx = SCALAR_ADD(tx, t30);
	SCALAR_STORE((temp + 15 * blockDim.x + threadIdx.x), tx);
      }

      if(threadIdx.x < npts - p_outer) {
	double *Xik = (Xi + p_outer + p_inner);
	double *Xjk = (Xj + p_outer + p_inner);
	double *Gik = (Gi + p_outer + p_inner);
	double *Gjk = (Gj + p_outer + p_inner);

	SCALAR_TYPE const_value_v = SCALAR_LOAD((weights + p_outer + p_inner));

	double const_value, X_ABp, Y_ABp, Z_ABp, comb_m_i, comb_n_j, comb_p_k;
	SCALAR_TYPE const_value_w;
	SCALAR_TYPE tx, ty, tz, tw, t0, t1, t2, t3, t4, t5;

	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t0 = SCALAR_LOAD((temp + 6 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	SCALAR_STORE((Gik + 0 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t1 = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	SCALAR_STORE((Gik + 1 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t2 = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	SCALAR_STORE((Gik + 2 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t3 = SCALAR_LOAD((temp + 9 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	SCALAR_STORE((Gik + 3 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t4 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	SCALAR_STORE((Gik + 4 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t5 = SCALAR_LOAD((temp + 11 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	SCALAR_STORE((Gik + 5 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	X_ABp = SCALAR_MUL(X_ABp, X_AB); comb_m_i = SCALAR_MUL(comb_m_i * 1, SCALAR_RECIPROCAL(1));
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	SCALAR_STORE((Gik + 0 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	SCALAR_STORE((Gik + 1 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	SCALAR_STORE((Gik + 2 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t3 = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	SCALAR_STORE((Gik + 3 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t4 = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	SCALAR_STORE((Gik + 4 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	t5 = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	SCALAR_STORE((Gik + 5 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t0 = SCALAR_LOAD((temp + 7 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	SCALAR_STORE((Gik + 0 * ldG), tz);
	SCALAR_STORE((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t1 = SCALAR_LOAD((temp + 9 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	SCALAR_STORE((Gik + 1 * ldG), tz);
	SCALAR_STORE((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t2 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	SCALAR_STORE((Gik + 2 * ldG), tz);
	SCALAR_STORE((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t3 = SCALAR_LOAD((temp + 12 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	SCALAR_STORE((Gik + 3 * ldG), tz);
	SCALAR_STORE((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t4 = SCALAR_LOAD((temp + 13 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	SCALAR_STORE((Gik + 4 * ldG), tz);
	SCALAR_STORE((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t5 = SCALAR_LOAD((temp + 14 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	SCALAR_STORE((Gik + 5 * ldG), tz);
	SCALAR_STORE((Gjk + 1 * ldG), tw);
	Y_ABp = SCALAR_MUL(Y_ABp, Y_AB); comb_n_j = SCALAR_MUL(comb_n_j * 1, SCALAR_RECIPROCAL(1));
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	SCALAR_STORE((Gik + 0 * ldG), tz);
	SCALAR_STORE((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	SCALAR_STORE((Gik + 1 * ldG), tz);
	SCALAR_STORE((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	SCALAR_STORE((Gik + 2 * ldG), tz);
	SCALAR_STORE((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t3 = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	SCALAR_STORE((Gik + 3 * ldG), tz);
	SCALAR_STORE((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t4 = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	SCALAR_STORE((Gik + 4 * ldG), tz);
	SCALAR_STORE((Gjk + 1 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 1 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 1 * ldG));
	t5 = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	SCALAR_STORE((Gik + 5 * ldG), tz);
	SCALAR_STORE((Gjk + 1 * ldG), tw);
	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t0 = SCALAR_LOAD((temp + 8 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	SCALAR_STORE((Gik + 0 * ldG), tz);
	SCALAR_STORE((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t1 = SCALAR_LOAD((temp + 10 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	SCALAR_STORE((Gik + 1 * ldG), tz);
	SCALAR_STORE((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t2 = SCALAR_LOAD((temp + 11 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	SCALAR_STORE((Gik + 2 * ldG), tz);
	SCALAR_STORE((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t3 = SCALAR_LOAD((temp + 13 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	SCALAR_STORE((Gik + 3 * ldG), tz);
	SCALAR_STORE((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t4 = SCALAR_LOAD((temp + 14 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	SCALAR_STORE((Gik + 4 * ldG), tz);
	SCALAR_STORE((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t5 = SCALAR_LOAD((temp + 15 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	SCALAR_STORE((Gik + 5 * ldG), tz);
	SCALAR_STORE((Gjk + 2 * ldG), tw);
	Z_ABp = SCALAR_MUL(Z_ABp, Z_AB); comb_p_k = SCALAR_MUL(comb_p_k * 1, SCALAR_RECIPROCAL(1));
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	SCALAR_STORE((Gik + 0 * ldG), tz);
	SCALAR_STORE((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 1 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 1 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t1 = SCALAR_LOAD((temp + 1 * blockDim.x + threadIdx.x));
	t1 = SCALAR_MUL(t1, const_value_w);
	tz = SCALAR_FMA(ty, t1, tz);
	tw = SCALAR_FMA(tx, t1, tw);
	SCALAR_STORE((Gik + 1 * ldG), tz);
	SCALAR_STORE((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 2 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 2 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t2 = SCALAR_LOAD((temp + 2 * blockDim.x + threadIdx.x));
	t2 = SCALAR_MUL(t2, const_value_w);
	tz = SCALAR_FMA(ty, t2, tz);
	tw = SCALAR_FMA(tx, t2, tw);
	SCALAR_STORE((Gik + 2 * ldG), tz);
	SCALAR_STORE((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 3 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 3 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t3 = SCALAR_LOAD((temp + 3 * blockDim.x + threadIdx.x));
	t3 = SCALAR_MUL(t3, const_value_w);
	tz = SCALAR_FMA(ty, t3, tz);
	tw = SCALAR_FMA(tx, t3, tw);
	SCALAR_STORE((Gik + 3 * ldG), tz);
	SCALAR_STORE((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 4 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 4 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t4 = SCALAR_LOAD((temp + 4 * blockDim.x + threadIdx.x));
	t4 = SCALAR_MUL(t4, const_value_w);
	tz = SCALAR_FMA(ty, t4, tz);
	tw = SCALAR_FMA(tx, t4, tw);
	SCALAR_STORE((Gik + 4 * ldG), tz);
	SCALAR_STORE((Gjk + 2 * ldG), tw);
	tx = SCALAR_LOAD((Xik + 5 * ldX));
	ty = SCALAR_LOAD((Xjk + 2 * ldX));
	tz = SCALAR_LOAD((Gik + 5 * ldG));
	tw = SCALAR_LOAD((Gjk + 2 * ldG));
	t5 = SCALAR_LOAD((temp + 5 * blockDim.x + threadIdx.x));
	t5 = SCALAR_MUL(t5, const_value_w);
	tz = SCALAR_FMA(ty, t5, tz);
	tw = SCALAR_FMA(tx, t5, tw);
	SCALAR_STORE((Gik + 5 * ldG), tz);
	SCALAR_STORE((Gjk + 2 * ldG), tw);
      }
    }
  }

  void integral_2_1(double X_AB,
		    double Y_AB,
		    double Z_AB,
		    size_t npts,
		    double *points,
        shell_pair* sp,
		    double *Xi,
		    double *Xj,
		    int ldX,
		    double *Gi,
		    double *Gj,
		    int ldG, 
		    double *weights, 
		    double *boys_table) {
    dev_integral_2_1<<<320, 128>>>(X_AB,
				   Y_AB,
				   Z_AB,
				   npts,
				   points,
           sp,
				   Xi,
				   Xj,
				   ldX,
				   Gi,
				   Gj,
				   ldG, 
				   weights,
				   boys_table);
  }
}
