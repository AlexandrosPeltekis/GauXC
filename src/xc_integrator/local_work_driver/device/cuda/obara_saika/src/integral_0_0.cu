#include "hip/hip_runtime.h"
#include <math.h>
#include "../include/gpu/chebyshev_boys_computation.hpp"
#include "../include/gpu/integral_data_types.hpp"
#include "config_obara_saika.hpp"
#include "integral_0_0.hu"
#include <gauxc/util/div_ceil.hpp>
#include "device/xc_device_task.hpp"
#include "../../../common/shell_pair_to_task.hpp"

#define PI 3.14159265358979323846

#define MIN(a,b)				\
  ({ __typeof__ (a) _a = (a);			\
    __typeof__ (b) _b = (b);			\
    _a < _b ? _a : _b; })

namespace XGPU {
  __inline__ __device__ void dev_integral_0_0_driver(size_t npts,
				   double *points_x,
				   double *points_y,
				   double *points_z,
           shell_pair* sp,
				   double *Xi,
				   double *Xj,
				   int ldX,
				   double *Gi,
				   double *Gj,
				   int ldG, 
				   double *weights, 
				   double *boys_table) {
    //__shared__ double temp[128 * 1];
    double temp;
    const auto nprim_pairs = sp->nprim_pairs();
    const auto prim_pairs  = sp->prim_pairs();
    
    for(size_t p_outer = blockIdx.x * blockDim.x; p_outer < npts; p_outer += gridDim.x * blockDim.x) {
      double *_point_outer_x = (points_x + p_outer);
      double *_point_outer_y = (points_y + p_outer);
      double *_point_outer_z = (points_z + p_outer);

      size_t p_inner = (threadIdx.x < (npts - p_outer)) ? threadIdx.x : (npts - p_outer);

      //for(int i = 0; i < 1; ++i) SCALAR_STORE((temp + i * blockDim.x + threadIdx.x), SCALAR_ZERO());
      temp = SCALAR_ZERO();

      for(int ij = 0; ij < nprim_pairs; ++ij) {
	double RHO = prim_pairs[ij].gamma;

	double xP = prim_pairs[ij].P.x;
	double yP = prim_pairs[ij].P.y;
	double zP = prim_pairs[ij].P.z;

	double eval = prim_pairs[ij].K_coeff_prod;

	// Evaluate T Values
	SCALAR_TYPE xC = SCALAR_LOAD((_point_outer_x + p_inner));
	SCALAR_TYPE yC = SCALAR_LOAD((_point_outer_y + p_inner));
	SCALAR_TYPE zC = SCALAR_LOAD((_point_outer_z + p_inner));

	SCALAR_TYPE X_PC = SCALAR_SUB(xP, xC);
	SCALAR_TYPE Y_PC = SCALAR_SUB(yP, yC);
	SCALAR_TYPE Z_PC = SCALAR_SUB(zP, zC);

	SCALAR_TYPE TVAL = SCALAR_MUL(X_PC, X_PC);
	TVAL = SCALAR_FMA(Y_PC, Y_PC, TVAL);
	TVAL = SCALAR_FMA(Z_PC, Z_PC, TVAL);
	TVAL = SCALAR_MUL(RHO, TVAL);

	SCALAR_TYPE t00, TVAL_inv_e;

	// Evaluate Boys function
	boys_element<0>(&TVAL, &TVAL_inv_e, &t00, boys_table);

	// Evaluate VRR Buffer
	SCALAR_TYPE tx;


	t00 = SCALAR_MUL(eval, t00);
	//tx = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
	tx = temp;
	tx = SCALAR_ADD(tx, t00);
	//SCALAR_STORE((temp + 0 * blockDim.x + threadIdx.x), tx);
  temp = tx;
      }

      if(threadIdx.x < npts - p_outer) {
	double *Xik = (Xi + p_outer + p_inner);
	double *Xjk = (Xj + p_outer + p_inner);
	double *Gik = (Gi + p_outer + p_inner);
	double *Gjk = (Gj + p_outer + p_inner);

	SCALAR_TYPE const_value_v = SCALAR_LOAD((weights + p_outer + p_inner));

	double const_value, X_ABp, Y_ABp, Z_ABp, comb_m_i, comb_n_j, comb_p_k;
	SCALAR_TYPE const_value_w;
	SCALAR_TYPE tx, ty, tz, tw, t0;

	X_ABp = 1.0; comb_m_i = 1.0;
	Y_ABp = 1.0; comb_n_j = 1.0;
	Z_ABp = 1.0; comb_p_k = 1.0;
	const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
	const_value_w = SCALAR_MUL(const_value_v, const_value);
	tx = SCALAR_LOAD((Xik + 0 * ldX));
	ty = SCALAR_LOAD((Xjk + 0 * ldX));
  #if 0
	tz = SCALAR_LOAD((Gik + 0 * ldG));
	tw = SCALAR_LOAD((Gjk + 0 * ldG));
	//t0 = SCALAR_LOAD((temp + 0 * blockDim.x + threadIdx.x));
  t0 = temp;
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_FMA(ty, t0, tz);
	tw = SCALAR_FMA(tx, t0, tw);
	SCALAR_STORE((Gik + 0 * ldG), tz);
	SCALAR_STORE((Gjk + 0 * ldG), tw);
  #else
  t0 = temp;
	t0 = SCALAR_MUL(t0, const_value_w);
	tz = SCALAR_MUL(ty, t0);
	tw = SCALAR_MUL(tx, t0);
  atomicAdd(Gik, tz);
  atomicAdd(Gjk, tw);
  #endif
      }
    }
  }





  __global__ void dev_integral_0_0(size_t npts,
				   double *points_x,
				   double *points_y,
				   double *points_z,
           shell_pair* sp,
				   double *Xi,
				   double *Xj,
				   int ldX,
				   double *Gi,
				   double *Gj,
				   int ldG, 
				   double *weights, 
				   double *boys_table) {
    dev_integral_0_0_driver( npts, points_x, points_y, points_z, sp, Xi, Xj, ldX,
      Gi, Gj, ldG, weights, boys_table );
  }



  void integral_0_0(size_t npts,
		    double *points_x,
		    double *points_y,
		    double *points_z,
        shell_pair* sp,
		    double *Xi,
		    double *Xj,
		    int ldX,
		    double *Gi,
		    double *Gj,
		    int ldG, 
		    double *weights, 
		    double *boys_table,
        hipStream_t stream) {
    int nthreads = 128;
    int nblocks = std::min(intmax_t(320), GauXC::util::div_ceil(npts,nthreads));
    dev_integral_0_0<<<nblocks, nthreads,0,stream>>>(npts,
				   points_x,
				   points_y,
				   points_z,
           sp,
				   Xi,
				   Xj,
				   ldX,
				   Gi,
				   Gj,
				   ldG, 
				   weights,
				   boys_table);
  }





  __global__ void dev_integral_0_0_batched(
           const GauXC::ShellPairToTaskDevice* sp2task,
           GauXC::XCDeviceTask*                device_tasks,
				   double *boys_table) {

    const int ntask = sp2task->ntask;
    for( int i_task = blockIdx.y; i_task < ntask; i_task += gridDim.y ) {
    
      const auto iT = sp2task->task_idx_device[i_task];
      const auto* task  = device_tasks + iT;
      const auto  npts  = task->npts;

      const auto  i_off = sp2task->task_shell_off_row_device[i_task]*npts;
      const auto  j_off = sp2task->task_shell_off_col_device[i_task]*npts;

      dev_integral_0_0_driver( 
        npts,
        task->points_x,
        task->points_y,
        task->points_z,
        sp2task->shell_pair_device,
        task->fmat + i_off,
        task->fmat + j_off,
        npts,
        task->gmat + i_off,
        task->gmat + j_off,
        npts,
        task->weights, boys_table );
    }

  }


  void integral_0_0_batched(size_t ntask_sp,
        const GauXC::ShellPairToTaskDevice* sp2task,
        GauXC::XCDeviceTask*                device_tasks,
		    double *boys_table,
        hipStream_t stream) {

    int nthreads = 128;
    int nblocks_x = 160;
    int nblocks_y = ntask_sp;
    dim3 nblocks(nblocks_x, nblocks_y);
    dev_integral_0_0_batched<<<nblocks,nthreads,0,stream>>>(
      sp2task, device_tasks, boys_table );

  }
}
