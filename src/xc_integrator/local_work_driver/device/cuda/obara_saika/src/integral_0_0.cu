#include "hip/hip_runtime.h"
#include <math.h>
#include "../include/gpu/chebyshev_boys_computation.hpp"
#include "config_obara_saika.hpp"
#include "integral_0_0.hu"

namespace XGPU {
  __inline__ __device__ void dev_integral_0_0_driver(size_t npts, 
				   const double *points_x,
				   const double *points_y,
				   const double *points_z,
           const shell_pair* sp,
				   const double *Xi,
				   const double *Xj,
				   int ldX,
				   double *Gi,
				   double *Gj,
				   int ldG, 
				   const double *weights, 
				   const double *boys_table) {

    double temp;

    // Load PrimPairs into shared mem
    const int nprim_pairs = sp->nprim_pairs();
    #if 1
    __shared__ GauXC::PrimitivePair<double> prim_pairs[GauXC::detail::nprim_pair_max];
    __syncthreads();
    if( threadIdx.x < 32 ) {
      const auto pp = sp->prim_pairs();
      for(int ij = threadIdx.x; ij < nprim_pairs; ij += 32) {
        prim_pairs[ij] = pp[ij];
      }
    }
    __syncthreads();
    #else
    const auto& prim_pairs = sp->prim_pairs();
    #endif

    const int npts_int = (int) npts;

    #pragma unroll(1)
    for(int p_outer = blockIdx.x * 128; p_outer < npts_int; p_outer += gridDim.x * 128) {

      const double * __restrict__ _point_outer_x = (points_x + p_outer);
      const double * __restrict__ _point_outer_y = (points_y + p_outer);
      const double * __restrict__ _point_outer_z = (points_z + p_outer);

      int p_inner = threadIdx.x;
      if (threadIdx.x < npts_int - p_outer) {

      temp = SCALAR_ZERO();
	    const SCALAR_TYPE xC = SCALAR_LOAD((_point_outer_x + p_inner));
	    const SCALAR_TYPE yC = SCALAR_LOAD((_point_outer_y + p_inner));
	    const SCALAR_TYPE zC = SCALAR_LOAD((_point_outer_z + p_inner));

      for(int ij = 0; ij < nprim_pairs; ++ij) {
        double RHO = prim_pairs[ij].gamma;
      
        double xP = prim_pairs[ij].P.x;
        double yP = prim_pairs[ij].P.y;
        double zP = prim_pairs[ij].P.z;
      
        double eval = prim_pairs[ij].K_coeff_prod;
      
        // Evaluate T Values
        const SCALAR_TYPE X_PC = SCALAR_SUB(xP, xC);
        const SCALAR_TYPE Y_PC = SCALAR_SUB(yP, yC);
        const SCALAR_TYPE Z_PC = SCALAR_SUB(zP, zC);
      
        SCALAR_TYPE TVAL = SCALAR_MUL(X_PC, X_PC);
        TVAL = SCALAR_FMA(Y_PC, Y_PC, TVAL);
        TVAL = SCALAR_FMA(Z_PC, Z_PC, TVAL);
        TVAL = SCALAR_MUL(RHO, TVAL);
      
        // Evaluate VRR Buffer
        const SCALAR_TYPE t00 = boys_element_0(TVAL);
        temp = SCALAR_FMA( eval, t00, temp );
      }
      if (abs(temp) > 1e-12) {
        const double * __restrict__ Xik = (Xi + p_outer + p_inner);
        const double * __restrict__ Xjk = (Xj + p_outer + p_inner);
        double * __restrict__ Gik = (Gi + p_outer + p_inner);
        double * __restrict__ Gjk = (Gj + p_outer + p_inner);
      
        SCALAR_TYPE const_value_v = SCALAR_LOAD((weights + p_outer + p_inner));
      
        double const_value, X_ABp, Y_ABp, Z_ABp, comb_m_i, comb_n_j, comb_p_k;
        SCALAR_TYPE const_value_w;
        SCALAR_TYPE tx, ty, tz, tw, t0;
      
        X_ABp = 1.0; comb_m_i = 1.0;
        Y_ABp = 1.0; comb_n_j = 1.0;
        Z_ABp = 1.0; comb_p_k = 1.0;
        const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
        const_value_w = SCALAR_MUL(const_value_v, const_value);
        tx = SCALAR_LOAD(Xik);
        ty = SCALAR_LOAD(Xjk);
        t0 = SCALAR_MUL(temp, const_value_w);
        tz = SCALAR_MUL(ty, t0);
        tw = SCALAR_MUL(tx, t0);
        atomicAdd(Gik, tz);
        atomicAdd(Gjk, tw);
      }
      }
    }
  }





  __global__ void dev_integral_0_0(size_t npts,
				   double *points_x,
				   double *points_y,
				   double *points_z,
           shell_pair* sp,
				   double *Xi,
				   double *Xj,
				   int ldX,
				   double *Gi,
				   double *Gj,
				   int ldG, 
				   double *weights, 
				   double *boys_table) {
    dev_integral_0_0_driver( npts, points_x, points_y, points_z, sp, Xi, Xj, ldX,
      Gi, Gj, ldG, weights, boys_table );
  }



  void integral_0_0(size_t npts,
		    double *points_x,
		    double *points_y,
		    double *points_z,
        shell_pair* sp,
		    double *Xi,
		    double *Xj,
		    int ldX,
		    double *Gi,
		    double *Gj,
		    int ldG, 
		    double *weights, 
		    double *boys_table,
        hipStream_t stream) {
    int nthreads = 128;
    int nblocks = std::min(intmax_t(320), GauXC::util::div_ceil(npts,nthreads));
    dev_integral_0_0<<<nblocks, nthreads,0,stream>>>(npts,
				   points_x,
				   points_y,
				   points_z,
           sp,
				   Xi,
				   Xj,
				   ldX,
				   Gi,
				   Gj,
				   ldG, 
				   weights,
				   boys_table);
  }





  __inline__ __device__ void dev_integral_0_0_batched_driver(
           const GauXC::ShellPairToTaskDevice* sp2task,
           GauXC::XCDeviceTask*                device_tasks,
				   double *boys_table) {

    //if (sp2task->shell_pair_device->nprim_pairs() == 0) return;
    const int ntask = sp2task->ntask;

    for( int i_task = blockIdx.y; i_task < ntask; i_task += gridDim.y ) {
    
      const auto iT = sp2task->task_idx_device[i_task];
      const auto* task  = device_tasks + iT;
      const auto  npts  = task->npts;

      const auto  i_off = sp2task->task_shell_off_row_device[i_task]*npts;
      const auto  j_off = sp2task->task_shell_off_col_device[i_task]*npts;

      dev_integral_0_0_driver( 
        npts,
        task->points_x,
        task->points_y,
        task->points_z,
        sp2task->shell_pair_device,
        task->fmat + i_off,
        task->fmat + j_off,
        npts,
        task->gmat + i_off,
        task->gmat + j_off,
        npts,
        task->weights, boys_table );
    }

  }

  __global__ void dev_integral_0_0_batched(
           const GauXC::ShellPairToTaskDevice* sp2task,
           GauXC::XCDeviceTask*                device_tasks,
				   double *boys_table) {
    dev_integral_0_0_batched_driver( sp2task, device_tasks, boys_table );
  }

  void integral_0_0_batched(size_t ntask_sp,
        const GauXC::ShellPairToTaskDevice* sp2task,
        GauXC::XCDeviceTask*                device_tasks,
		    double *boys_table,
        hipStream_t stream) {

    int nthreads = 128;
    int nblocks_x = 160;
    int nblocks_y = ntask_sp;
    dim3 nblocks(nblocks_x, nblocks_y);
    dev_integral_0_0_batched<<<nblocks,nthreads,0,stream>>>(
      sp2task, device_tasks, boys_table );

  }





  __inline__ __device__ void dev_integral_0_0_soa_batched_driver(
           int32_t                         ntask,
           const int32_t*                  sp2task_idx_device,
           const int32_t*                  sp2task_shell_off_row_device,
           const int32_t*                  sp2task_shell_off_col_device,
           const GauXC::ShellPair<double>* shell_pair_device,
           const int32_t*                  task_npts,
           const double**                  task_points_x,
           const double**                  task_points_y,
           const double**                  task_points_z,
           const double**                  task_weights,
           const double**                  task_fmat,
           double**                        task_gmat,
				   double *                        boys_table) {

    for( int i_task = blockIdx.y; i_task < ntask; i_task += gridDim.y ) {
    
      const auto iT   = sp2task_idx_device[i_task];
      const auto npts = task_npts[iT];

      const auto  i_off = sp2task_shell_off_row_device[i_task] * npts;
      const auto  j_off = sp2task_shell_off_col_device[i_task] * npts;

      dev_integral_0_0_driver( 
        npts,
        task_points_x[iT],
        task_points_y[iT],
        task_points_z[iT],
        shell_pair_device,
        task_fmat[iT] + i_off,
        task_fmat[iT] + j_off,
        npts,
        task_gmat[iT] + i_off,
        task_gmat[iT] + j_off,
        npts,
        task_weights[iT], boys_table );
    }

  }

  __global__ void dev_integral_0_0_soa_batched(
           int32_t                         ntask,
           const int32_t*                  sp2task_idx_device,
           const int32_t*                  sp2task_shell_off_row_device,
           const int32_t*                  sp2task_shell_off_col_device,
           const GauXC::ShellPair<double>* shell_pair_device,
           const int32_t*                  task_npts,
           const double**                   task_points_x,
           const double**                   task_points_y,
           const double**                   task_points_z,
           const double**                   task_weights,
           const double**                   task_fmat,
           double**                         task_gmat,
				   double *boys_table) {
    dev_integral_0_0_soa_batched_driver( ntask, sp2task_idx_device, 
      sp2task_shell_off_row_device, sp2task_shell_off_col_device, shell_pair_device,
      task_npts, task_points_x, task_points_y, task_points_z, task_weights,
      task_fmat, task_gmat, boys_table );
  }


  __global__ void 
  __launch_bounds__(128, 16)
  dev_integral_0_0_shell_batched(
           int nsp,
           const GauXC::ShellPairToTaskDevice* sp2task,
           GauXC::XCDeviceTask*                device_tasks,
				   double *boys_table) {

    for( int i = blockIdx.z; i < nsp; i += gridDim.z ) {
      dev_integral_0_0_batched_driver( sp2task + i, device_tasks, boys_table );
    }

  }

  void integral_0_0_shell_batched(
        size_t nsp,
        size_t max_ntask,
        const GauXC::ShellPairToTaskDevice* sp2task,
        GauXC::XCDeviceTask*                device_tasks,
		    double *boys_table,
        hipStream_t stream) {

    size_t xy_max = (1ul << 16) - 1;
    int nthreads = 128;
    int nblocks_x = 1;
    int nblocks_y = std::min(max_ntask, xy_max);
    int nblocks_z = std::min(nsp,  xy_max);
    dim3 nblocks(nblocks_x, nblocks_y, nblocks_z);

    dev_integral_0_0_shell_batched<<<nblocks,nthreads,0,stream>>>(
      nsp, sp2task, device_tasks, boys_table );

  }
   
#define K00_WARPSIZE 32
#define K00_NUMWARPS 8
#define K00_NUMTHREADS (K00_WARPSIZE * K00_NUMWARPS)
#define K00_USESHARED 1
#define K00_MAX_PRIMPAIRS 32

__inline__ __device__ void dev_integral_0_0_task(
  const int i,
  const int npts,
  const int nprim_pairs,
  // Point data
  double4 (&s_task_data)[K00_NUMTHREADS],
  // Shell Pair Data
  const shell_pair* sp,
  // Output Data
  const double *Xi,
  const double *Xj,
  int ldX,
  double *Gi,
  double *Gj,
  int ldG, 
  // Other
  const double *boys_table) {


  const auto& prim_pairs = sp->prim_pairs();

  const int laneId = threadIdx.x % K00_WARPSIZE;

#if K00_USESHARED
  // Load Primpairs to shared
  __shared__ GauXC::PrimitivePair<double> s_prim_pairs[K00_NUMWARPS][K00_MAX_PRIMPAIRS];

  const int warpId = (threadIdx.x / K00_WARPSIZE);
  const int32_t* src = (int32_t*) &(prim_pairs[0]);
  int32_t* dst = (int32_t*) &(s_prim_pairs[warpId][0]);

  for (int i = laneId; i < nprim_pairs * sizeof(GauXC::PrimitivePair<double>) / sizeof(int32_t); i+=K00_WARPSIZE) {
    dst[i] = src[i]; 
  }
  __syncwarp();
#endif

  // Loop over points in shared in batches of 32
  for (int i = 0; i < K00_NUMTHREADS / K00_WARPSIZE; i++) {
    double temp = SCALAR_ZERO();

    const int pointIndex = i * K00_WARPSIZE + laneId;

    if (pointIndex < npts) {

      const double point_x = s_task_data[pointIndex].x;
      const double point_y = s_task_data[pointIndex].y;
      const double point_z = s_task_data[pointIndex].z;
      const double weight = s_task_data[pointIndex].w;

      for (int ij = 0; ij < nprim_pairs; ij++) {
#if K00_USESHARED
        double RHO = s_prim_pairs[warpId][ij].gamma;
        double xP = s_prim_pairs[warpId][ij].P.x;
        double yP = s_prim_pairs[warpId][ij].P.y;
        double zP = s_prim_pairs[warpId][ij].P.z;
        double eval = s_prim_pairs[warpId][ij].K_coeff_prod;
#else
        double RHO = prim_pairs[ij].gamma;
        double xP = prim_pairs[ij].P.x;
        double yP = prim_pairs[ij].P.y;
        double zP = prim_pairs[ij].P.z;
        double eval = prim_pairs[ij].K_coeff_prod;
#endif
      
        // Evaluate T Values
        const SCALAR_TYPE X_PC = SCALAR_SUB(xP, point_x);
        const SCALAR_TYPE Y_PC = SCALAR_SUB(yP, point_y);
        const SCALAR_TYPE Z_PC = SCALAR_SUB(zP, point_z);
      
        SCALAR_TYPE TVAL = SCALAR_MUL(X_PC, X_PC);
        TVAL = SCALAR_FMA(Y_PC, Y_PC, TVAL);
        TVAL = SCALAR_FMA(Z_PC, Z_PC, TVAL);
        TVAL = SCALAR_MUL(RHO, TVAL);
      
        // Evaluate VRR Buffer
        const SCALAR_TYPE t00 = boys_element_0(TVAL);
        temp = SCALAR_FMA( eval, t00, temp );
      }

      // Output
      if (abs(temp) > 1e-12) {
        const double * __restrict__ Xik = (Xi + pointIndex);
        const double * __restrict__ Xjk = (Xj + pointIndex);
        double * __restrict__ Gik = (Gi + pointIndex);
        double * __restrict__ Gjk = (Gj + pointIndex);

        SCALAR_TYPE const_value_v = weight;
      
        double const_value, X_ABp, Y_ABp, Z_ABp, comb_m_i, comb_n_j, comb_p_k;
        SCALAR_TYPE const_value_w;
        SCALAR_TYPE tx, ty, tz, tw, t0;
      
        X_ABp = 1.0; comb_m_i = 1.0;
        Y_ABp = 1.0; comb_n_j = 1.0;
        Z_ABp = 1.0; comb_p_k = 1.0;
        const_value = comb_m_i * comb_n_j * comb_p_k * X_ABp * Y_ABp * Z_ABp;
        const_value_w = SCALAR_MUL(const_value_v, const_value);
        tx = SCALAR_LOAD(Xik);
        ty = SCALAR_LOAD(Xjk);
        t0 = SCALAR_MUL(temp, const_value_w);
        tz = SCALAR_MUL(ty, t0);
        tw = SCALAR_MUL(tx, t0);
        atomicAdd(Gik, tz);
        atomicAdd(Gjk, tw);
      }
    }
  }
  __syncwarp();
}




__global__ void 
__launch_bounds__(K00_NUMTHREADS, 1)
dev_integral_0_0_task_batched(
  int ntask, int nsubtask,
  GauXC::XCDeviceTask*                device_tasks,
  const GauXC::TaskToShellPairDevice* task2sp,
  const int4* subtasks,
  const int32_t* nprim_pairs_device,
  shell_pair** sp_ptr_device,
  double *boys_table) {

  __shared__ double4 s_task_data[K00_NUMTHREADS];

  const int warpId = threadIdx.x / K00_WARPSIZE;
  
  const int i_subtask = blockIdx.x;
  const int i_task = subtasks[i_subtask].x;
  const int point_start = subtasks[i_subtask].y;
  const int point_end = subtasks[i_subtask].z;
  const int point_count = point_end - point_start;

  const auto* task = device_tasks + i_task;

  const int npts = task->npts;

  const auto* points_x = task->points_x;
  const auto* points_y = task->points_y;
  const auto* points_z = task->points_z;
  const auto* weights = task->weights;

  const auto nsp = task2sp[i_task].nsp;


  const int npts_block = (point_count + blockDim.x - 1) / blockDim.x;

  for (int i_block = 0; i_block < npts_block; i_block++) {
    const int i = point_start + i_block * blockDim.x;

    // load point into registers
    const double point_x = points_x[i + threadIdx.x];
    const double point_y = points_y[i + threadIdx.x];
    const double point_z = points_z[i + threadIdx.x];
    const double weight = weights[i + threadIdx.x];

    s_task_data[threadIdx.x].x = point_x;
    s_task_data[threadIdx.x].y = point_y;
    s_task_data[threadIdx.x].z = point_z;
    s_task_data[threadIdx.x].w = weight;
    __syncthreads();

    for (int j = K00_NUMWARPS*blockIdx.y+warpId; j < nsp; j+=K00_NUMWARPS*gridDim.y) {
      const auto i_off = task2sp[i_task].task_shell_off_row_device[j];
      const auto j_off = task2sp[i_task].task_shell_off_col_device[j];

      const auto index =  task2sp[i_task].shell_pair_linear_idx_device[j];
      const auto* sp = sp_ptr_device[index];
      const auto nprim_pairs = nprim_pairs_device[index];

      dev_integral_0_0_task(
        i, point_count, nprim_pairs,
        s_task_data,
        sp,
        task->fmat + i_off + i,
        task->fmat + j_off + i,
        npts,
        task->gmat + i_off + i,
        task->gmat + j_off + i,
        npts,
        boys_table);
    }
    __syncthreads();
  }
}

  void integral_0_0_task_batched(
    size_t ntasks, size_t nsubtask,
    size_t max_nsp,
    GauXC::XCDeviceTask*                device_tasks,
    const GauXC::TaskToShellPairDevice* task2sp,
    const std::array<int32_t, 4>*  subtasks,
    const int32_t* nprim_pairs_device,
    shell_pair** sp_ptr_device,
    double* sp_X_AB_device,
    double* sp_Y_AB_device,
    double* sp_Z_AB_device,
    double *boys_table,
    hipStream_t stream) {

    size_t xy_max = (1ul << 16) - 1;
    int nthreads = K00_NUMTHREADS;
    int nblocks_x = nsubtask;
    int nblocks_y = 8; //std::min(max_nsp,  xy_max);
    int nblocks_z = 1;
    dim3 nblocks(nblocks_x, nblocks_y, nblocks_z);

    dev_integral_0_0_task_batched<<<nblocks, nthreads, 0, stream>>>(
      ntasks, nsubtask,
      device_tasks, task2sp, 
      (int4*) subtasks, nprim_pairs_device, sp_ptr_device,
      boys_table );

  }

}
